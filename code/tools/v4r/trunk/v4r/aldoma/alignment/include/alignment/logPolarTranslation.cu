#include "hip/hip_runtime.h"
// includes, project
#include <stdio.h>
#include <cutil_inline.h>
#include <hipfft/hipfft.h>

//define kernel...
//needs to multiply img_1 by img_2 in complex stuff
//perform the inverse of the cross spectrum... we can't do it in device code...

typedef float2 Complex;

__global__ void
cross_power_multiply (hipfftComplex *img_1, hipfftComplex * img_2, int size_img, int width)
{
  float threshold = 15 * 15;
  float tmp_r, tmp_c, tmp;
  tmp_r = tmp_c = 0;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int i2 = (i % size_img);

  //low pass filters
  int row = i2 / width;
  int col = i2 % width;
  if ((row * row + col * col) <= threshold)
  {
    tmp_r = (img_2[i2].x * img_1[i].x) - (-img_2[i2].y * (img_1[i].y));
    tmp_c = (img_2[i2].x * (img_1[i].y)) + (-img_2[i2].y * img_1[i].x);
    tmp = sqrt (tmp_r * tmp_r + tmp_c * tmp_c);
    tmp_r /= tmp;
    tmp_c /= tmp;
  }

  img_1[i].x = tmp_r;
  img_1[i].y = tmp_c;
}

__global__ void
normalize (hipfftComplex *img_1, int scale_limit, int width, int n_elements_img2) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_image = (i % (n_elements_img2));
  if ( (idx_image % width) <= (size_t)scale_limit || (idx_image % width) >= (size_t) (width - scale_limit) )
  {
    img_1[i].x /= 193.0;
  }
  else
  {
    img_1[i].x = 0;
  }
}


extern "C"
void computeCrossCorrelations(Complex * img1, int size_img1, Complex * img2, int size_img2, int size_img, int SCALE_RANGE)
{
  //allocate memory in device

  int n_imgs = size_img1 / size_img;
  int width = sqrt(size_img / sizeof(Complex));
  int n_elements_img1 = size_img1 / sizeof(Complex);
  int n_elements_img2 = width * width;

  ////printf("Inside of computeCrossCorrelations, number of images= %d %d %d %d\n", n_imgs, width, n_elements_img1, size_img1);

  Complex *d_img1, *d_img2;

  hipMalloc((void**) &d_img1, sizeof(Complex) * n_elements_img1);
  //printf("Error alloc 1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_img1, img1, sizeof(Complex) * n_elements_img1, hipMemcpyHostToDevice);
  //printf("Error memcpy 1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void**) &d_img2, sizeof(Complex) * n_elements_img2);
  //printf("Error alloc 2: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_img2, img2, sizeof(Complex) * n_elements_img2, hipMemcpyHostToDevice);
  //printf("Error memcpy 2: %s\n", hipGetErrorString(hipGetLastError()));
  //Create FFT plans
  hipfftHandle plan_1, plan_2;
  hipfftPlan1d(&plan_1, n_elements_img2, HIPFFT_C2C, n_imgs);
  //printf("Error hipfftPlan1d 1: %s\n", hipGetErrorString(hipGetLastError()));
  hipfftPlan1d(&plan_2, n_elements_img2, HIPFFT_C2C, 1);
  //printf("Error hipfftPlan1d 2: %s\n", hipGetErrorString(hipGetLastError()));

  //Transform in place!
  hipfftExecC2C(plan_1, (hipfftComplex *)d_img1, (hipfftComplex *)d_img1, HIPFFT_FORWARD);
  //printf("Error plan 1: %s\n", hipGetErrorString(hipGetLastError()));
  hipfftExecC2C(plan_2, (hipfftComplex *)d_img2, (hipfftComplex *)d_img2, HIPFFT_FORWARD);
  //printf("Error plan 2: %s\n", hipGetErrorString(hipGetLastError()));

  //call kernel that will compute the cross power multiplication between one chunk of data_1 and data_2
  int N_threads = 512;
  int N_blocks = n_elements_img1 / N_threads;
  ////printf("blocks: %d threads: %d", N_blocks, N_threads);

  cross_power_multiply <<< N_blocks , N_threads >>> (d_img1, d_img2, n_elements_img2, width);
  //printf("cross_power_multiply: %s\n", hipGetErrorString(hipGetLastError()));

  hipfftExecC2C(plan_1, (hipfftComplex *)d_img1, (hipfftComplex *)d_img1, HIPFFT_BACKWARD);
  //printf("Error plan 1 inverse: %s\n", hipGetErrorString(hipGetLastError()));

  // Destroy the CUFFT plan.
  hipfftDestroy(plan_1);
  //printf("Error hipfftDestroy 1: %s\n", hipGetErrorString(hipGetLastError()));
  hipfftDestroy(plan_2);
  //printf("Error hipfftDestroy 2: %s\n", hipGetErrorString(hipGetLastError()));


  normalize <<< N_blocks , N_threads >>> (d_img1, SCALE_RANGE, width, n_elements_img2);
  //printf("Error normalize: %s\n", hipGetErrorString(hipGetLastError()));

  hipMemcpy(img1, d_img1, n_elements_img1 * sizeof(Complex), hipMemcpyDeviceToHost);
  //printf("Error hipMemcpyDeviceToHost: %s\n", hipGetErrorString(hipGetLastError()));

  //free cuda memory
  hipFree(d_img1);
  //printf("Error hipFree 1: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_img2);
  //printf("Error hipFree 2: %s\n", hipGetErrorString(hipGetLastError()));

}
