#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "align.h"

#include "census.h"



unsigned int g_Width, g_Height;
unsigned int g_disp_min, g_disp_max, g_disp_step;
unsigned int g_blockSize;
bool g_sparse;
int g_streamNr;

// CUDA arrays
hipArray *cuImageLeft, *cuImageRight;

// CUDA streams
hipStream_t streamImageLeft[2], streamImageRight[2], streamDM[2];

// Textures
texture<unsigned char, 2, hipReadModeElementType> texImageLeft;
texture<unsigned char, 2, hipReadModeElementType> texImageRight;
texture<unsigned int, 1, hipReadModeElementType> texDSI1d;
texture<unsigned int, 1, hipReadModeElementType> texDSITemp1d;

// Host-side page-locked memory
unsigned char *h_left[2], *h_right[2], *h_dm[2];

// device data pointer
unsigned int *d_DSI;
unsigned int *d_DSI_Temp;
unsigned int *d_censusLeft, *d_censusRight;
int *d_Costs, *d_Costs_LR, *d_Costs_RL;
float *d_DMI, *d_DMI2, *d_DMI_LR, *d_DMI_RL;




float g_CensusTiming[6];


#define PRINT_BANDWIDTH 0
#define USE_FAST_AGGREGATION 1

// 24 bit multiplication takes only 4 clock cycles compared to 16 clock 
// cycles for the normal multiplication
#define USE_24BIT_MULTIPLICATION 1

#if USE_24BIT_MULTIPLICATION
	#define IMUL(a, b) __mul24(a, b)
#else
	#define IMUL(a, b) (a * b)
#endif

#define DATA_SIZE (g_Width*g_Height*sizeof(unsigned char))

#define CENSUS_RADIUS 8
#define CENSUS_TILE_W 64
#define CENSUS_TILE_H 64
#define CENSUS_SMEM_W (2 * CENSUS_RADIUS + CENSUS_TILE_W)
#define CENSUS_SMEM_H (2 * CENSUS_RADIUS + CENSUS_TILE_H)

#define CENSUS_SPARSE_RADIUS 8
#define CENSUS_SPARSE_TILE_W 64
#define CENSUS_SPARSE_TILE_H 32
#define CENSUS_SPARSE_SMEM_W (2 * CENSUS_SPARSE_RADIUS + CENSUS_SPARSE_TILE_W)
#define CENSUS_SPARSE_SMEM_H (2 * CENSUS_SPARSE_RADIUS + CENSUS_SPARSE_TILE_H)

#define START_TIMER \
		unsigned int hTimer; \
		CUT_SAFE_CALL(cutCreateTimer(&hTimer)); \
		CUDA_SAFE_CALL( hipDeviceSynchronize() ); \
		CUT_SAFE_CALL( cutResetTimer(hTimer) ); \
		CUT_SAFE_CALL( cutStartTimer(hTimer) );

#define STOP_TIMER(gpuTime) \
		CUDA_SAFE_CALL( hipDeviceSynchronize() ); \
		CUT_SAFE_CALL(cutStopTimer(hTimer)); \
		gpuTime = cutGetTimerValue(hTimer);


void setCensusTiming(CensusStep s, float timeMs) {
	g_CensusTiming[s] = timeMs;
}

extern "C" float getCensusTiming(CensusStep s) {
	return g_CensusTiming[s];
}

extern "C" unsigned int getCensusFLOP(CensusStep s) {
	int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;

	switch (s) {
		case eAggregateCosts:
			return disparities*60*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		case eCalcDSI:
			return (5*(128+disparities)*2*g_Height*iDivUp(g_Width, 128)+(17+43*disparities)*g_Width*g_Height);
		
		case eCensusTransform:
			return 2*3000*16*16*iDivUp(g_Width, 64)*iDivUp(g_Height, 64);
		
		case eCompareDisps:
			return 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		case eRefineSubPixel:
			return ((disparities*7+35)*g_Width*g_Height) + ((disparities*7+43)*g_Width*g_Height);
		
		case eRoundAndScaleDisparities:
			return 12*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		default:
			return 0;
	}
}

extern "C" unsigned int getCensusMemory(CensusStep s) {
	int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;

	switch (s) {
		case eAggregateCosts:
			return disparities*(g_Width*g_Height*sizeof(int) + iDivUp(g_Width, 16)*iDivUp(g_Height, 16)*21*21*sizeof(int));
		
		case eCalcDSI:
			return (iDivUp(g_Width, 128)*g_Height*2*(128 + disparities)*2*sizeof(unsigned int)+2*(g_Width-g_disp_max)*g_Height*sizeof(unsigned int));
		
		case eCensusTransform:
			return 2*(iDivUp(g_Width, CENSUS_SPARSE_TILE_W)*iDivUp(g_Height, CENSUS_SPARSE_TILE_H)*CENSUS_SPARSE_SMEM_W*CENSUS_SMEM_H*sizeof(unsigned int)+g_Width*g_Height*8);
		
		case eCompareDisps:
			return 2*g_Width*g_Height*sizeof(float);
		
		case eRefineSubPixel:
			return 2*(disparities+2)*g_Width*g_Height*sizeof(int);
		
		case eRoundAndScaleDisparities:
			return 2*g_Width*g_Height*sizeof(float);
		
		default:
			return 0;
	}
}


inline
__device__ int f2i(float f) {

	return f < 0 ? f - .5 : f + .5; 

}

template <class T>
inline
__device__ T getArrayValue(const T *arr, int x, int y, int width) {

	return arr[IMUL(y, width) + x];

}

template <class T>
inline
__device__ T getArrayValue(const T *arr, int x, int y, int z, int width, int height) {

	return arr[z * IMUL(width, height) + IMUL(y, width) + x];

}

template <class T>
inline
__device__ void setArrayValue(T *arr, T value, int x, int y, int width) {

	arr[IMUL(y, width) + x] = value;

}

template <class T>
inline
__device__ void setArrayValue(T *arr, T value, int x, int y, int z, int width, int height) {

	arr[z * IMUL(width, height) + IMUL(y, width) + x] = value;

}

template <bool sparse>
inline
__device__ unsigned char censusRow(unsigned char ref, unsigned char *row) {
	unsigned char c = 0;

	if (sparse) {
		if (ref > row[ 0]) c |= (1 << 7);
		if (ref > row[ 2]) c |= (1 << 6);
		if (ref > row[ 4]) c |= (1 << 5);
		if (ref > row[ 6]) c |= (1 << 4);
		if (ref > row[ 8]) c |= (1 << 3);
		if (ref > row[10]) c |= (1 << 2);
		if (ref > row[12]) c |= (1 << 1);
		if (ref > row[14]) c |= (1 << 0);
	}

	return c;
}

template <bool left>
__global__ void censusTransformSparse(unsigned int *d_census, int iWidth, int iHeight) {
	const int blockX = IMUL(blockIdx.x, CENSUS_SPARSE_TILE_W);
	const int blockY = IMUL(blockIdx.y, CENSUS_SPARSE_TILE_H);
	int x, y;

	__shared__ unsigned char smTileData[CENSUS_SPARSE_SMEM_H][CENSUS_SPARSE_SMEM_W];

	for (y = threadIdx.y; y < CENSUS_SPARSE_SMEM_H; y+=blockDim.y) { // 130 FLOP
		for (x = threadIdx.x; x < CENSUS_SPARSE_SMEM_W; x+=blockDim.x) {
			if (left)
				smTileData[y][x] = tex2D(texImageLeft, blockX+x-CENSUS_SPARSE_RADIUS, blockY+y-CENSUS_SPARSE_RADIUS);
			else
				smTileData[y][x] = tex2D(texImageRight, blockX+x-CENSUS_SPARSE_RADIUS, blockY+y-CENSUS_SPARSE_RADIUS);
		}
	}


	// assure that shared memory is completely loaded
	__syncthreads();


	// start calculation
	int2 c;
	unsigned char p1;
	int xref, yref;
	int idx = 0;

	//for (yref = threadIdx.y * 4; yref < threadIdx.y * 4 + 4 && yref < CENSUS_SPARSE_TILE_H && blockY+yref < iHeight; yref++) { // 2868 FLOP
	for (yref = threadIdx.y * 2; yref < threadIdx.y * 2 + 2 && yref < CENSUS_SPARSE_TILE_H && blockY+yref < iHeight; yref++) { // 2868 FLOP
		for (xref = threadIdx.x * 4; xref < threadIdx.x * 4 + 4 && xref < CENSUS_SPARSE_TILE_W && blockX+xref < iWidth; xref++) {
			p1 = smTileData[yref+CENSUS_SPARSE_RADIUS][xref+CENSUS_SPARSE_RADIUS];
			idx = (IMUL((blockY+yref), iWidth)+(blockX+xref));

			c.x = (censusRow<true>(p1, &smTileData[yref-8+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) << 24) | \
				  (censusRow<true>(p1, &smTileData[yref-6+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) << 16) | \
				  (censusRow<true>(p1, &smTileData[yref-4+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) <<  8) | \
				  (censusRow<true>(p1, &smTileData[yref-2+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) <<  0);

			c.y = (censusRow<true>(p1, &smTileData[yref-0+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) << 24) | \
				  (censusRow<true>(p1, &smTileData[yref+2+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) << 16) | \
				  (censusRow<true>(p1, &smTileData[yref+4+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) <<  8) | \
				  (censusRow<true>(p1, &smTileData[yref+6+CENSUS_SPARSE_RADIUS][xref-8+CENSUS_SPARSE_RADIUS]) <<  0);

			((int2*)d_census)[idx] = c;
		}
	}
}

template <bool left>
__global__ void censusTransform(unsigned int *d_census, int iWidth, int iHeight) {

	const int blockX = IMUL(blockIdx.x, CENSUS_TILE_W);
	const int blockY = IMUL(blockIdx.y, CENSUS_TILE_H);
	int x, y;

	__shared__ unsigned char smTileData[CENSUS_SMEM_H][CENSUS_SMEM_W];

	for (y = threadIdx.y; y < CENSUS_SMEM_H; y+=blockDim.y) { // 130 FLOP
		for (x = threadIdx.x; x < CENSUS_SMEM_W; x+=blockDim.x) {
			if (left)
				smTileData[y][x] = tex2D(texImageLeft, blockX+x-CENSUS_RADIUS, blockY+y-CENSUS_RADIUS);
			else
				smTileData[y][x] = tex2D(texImageRight, blockX+x-CENSUS_RADIUS, blockY+y-CENSUS_RADIUS);
		}
	}


	// assure that shared memory is completely loaded
	__syncthreads();


	// start calculation
	unsigned int c;
	char p1;
	char p2;
	int xref, yref;
	int idx = 0;
	int i;

	for (yref = threadIdx.y * 4; yref < threadIdx.y * 4 + 4 && yref < CENSUS_SPARSE_TILE_H && blockY+yref < iHeight; yref++) { // 2868 FLOP
		for (xref = threadIdx.x * 4; xref < threadIdx.x * 4 + 4 && xref < CENSUS_SPARSE_TILE_W && blockX+xref < iWidth; xref++) {
			p1 = smTileData[yref+CENSUS_RADIUS][xref+CENSUS_RADIUS];
			idx = (IMUL((blockY+yref), iWidth)+(blockX+xref))*8;

			for (i = 0; i < 8; i++) {
				c = 0;

				for (y = -7+i*2; y <= -6+i*2; y++) {
					for (x = -7; x <= 8; x++) {
						p2 = smTileData[yref+y+CENSUS_RADIUS][xref+x+CENSUS_RADIUS];

						c <<= 1;

						if (p1 > p2)
							c |= 1;
					}
				}

				d_census[idx+i] = c;
			}
		}
	}
}

template <int nrOfInt>
inline
__device__ int calcHamming(unsigned int *a, unsigned int *b) {
	unsigned int exor;
	int result = 0, j;

	for (j = 0; j < nrOfInt; j++)
	{
		exor = a[j] ^ b[j];

		// counts the set bits
		exor = exor - ((exor >> 1) & 0x55555555);
		exor = (exor & 0x33333333) + ((exor >> 2) & 0x33333333);
		exor = ((exor + (exor >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;

		result += exor;
	}

	return result;
}

template <int nrOfInt>
inline
__device__ int calcHamming2(unsigned int *a, unsigned int *b) {
	unsigned int exor;
	int result = 0, j, i;

	for (j = 0; j < nrOfInt; j++)
	{
		exor = a[j] ^ b[j];

		for (i = 0; i < sizeof(unsigned int); i++) {
			result += exor & 0x1u;
			exor >>= 1;
		}
	}

	return result;
}

template <int nrOfInt>
__global__ void calcDSI(unsigned int *DSI, unsigned int *census_L, unsigned int *census_R, int disp_start, int disp_stop, int disp_step, int iWidth, int iHeight) {
	const int disparities = (disp_stop - disp_start + 1) / disp_step;
	const int smWidth = (blockDim.x + disparities) * nrOfInt;

	const int x = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	const int y = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
	int blockX = IMUL(blockIdx.x, blockDim.x) * nrOfInt;
	int offset;
	int index;
	int i;

	// ((2 * (#threads + maxdisparity)) * nrOfInt) int values must be allocated
	extern __shared__ unsigned int smCensusData[];
	
	unsigned int *smCensusL = smCensusData;
	unsigned int *smCensusR = &smCensusData[smWidth];

	offset = IMUL(y, iWidth) * nrOfInt + blockX;
	for (i = threadIdx.x; i < smWidth && i + blockX < iWidth * nrOfInt; i+= blockDim.x) { //5*(blockDim.x + disparities) * nrOfInt FLOPS
		smCensusL[i] = census_L[offset+i];
		smCensusR[i] = census_R[offset+i];
	}


	unsigned int cost;
	int offset2;
	
	__syncthreads();

	//Calculation
	if (x < iWidth && y < iHeight) {
		offset  = IMUL((iWidth), iHeight); // 1 DSI page
		offset2 = IMUL((iWidth), y) + x;   // index within DSI page

		for (i = disp_start; i < disparities + disp_start && x < iWidth-(disp_start+disp_step*i); i++) { //43 FLOP per cycle
			index = (i-disp_start) * offset + offset2;

			cost = calcHamming<nrOfInt>(&smCensusR[threadIdx.x * nrOfInt], &smCensusL[(threadIdx.x + i*disp_step + disp_start) * nrOfInt]);
			DSI[index] = cost;

			// synchronize so that no serialization occurs
			__syncthreads();
		}
	}
}


template <int c>
__device__ unsigned int sumValues(unsigned int* arr) {
	return sumValues<c-1>(arr) + arr[c-1];
}

template <>
__device__ unsigned int sumValues<0>(unsigned int* arr) {
	return 0;
}

template <bool computeCapability_1_2, int blockSize>
__global__ void aggregateFilter(unsigned int *d_imageOut, unsigned int *d_imageIn, int iWidth, int iHeight) {
	const int blockX = IMUL(blockDim.x, blockIdx.x);
	const int blockY = IMUL(blockDim.y, blockIdx.y);
	const int radius = blockSize/2;
	int x = blockX + threadIdx.x;
	int y = blockY + threadIdx.y;
	int i, j;
	unsigned int data;

	__shared__ unsigned int smImageBlock[16+blockSize][16+blockSize+1];


	if (x < iWidth && y < iHeight) {

		if (computeCapability_1_2) {
			/* // ~40 ms
			for (y = threadIdx.y - radius; y < 16 + radius; y+=blockDim.y) {
				for (x = threadIdx.x - radius; x < 16 + radius; x+=blockDim.x) {
					if (x < 0 || y < 0 || x + blockX >= iWidth || y + blockY >= iHeight)
						data = 0;
					else
						data = getArrayValue(d_imageIn, blockX+x, blockY+y, iWidth);

					smImageBlock[y+radius][x+radius] = data;
				}
			}*/

			// Bad performance due to uncoalesced load, because -radius leads to 
			// misalignment on devices with Compute Capability < 1.2

			//~24 ms
			for (j = 0; threadIdx.y + j < 16 + blockSize; j+=16) {
				for (i = 0; threadIdx.x + i < 16 + blockSize; i+=16) {
					x = blockX + i + threadIdx.x - radius;
					y = blockY + j + threadIdx.y - radius;

					if (x < 0 || y < 0 || x >= iWidth || y >= iHeight)
						data = 0;
					else
						data = getArrayValue(d_imageIn, x, y, iWidth);
						//data = tex1Dfetch(texInt, y*iWidth+x);

					smImageBlock[j+threadIdx.y][i+threadIdx.x] = data;
					

					//smImageBlock[j+threadIdx.y][i+threadIdx.x] = tex2D(texDSI, x, y);
				}
			}

			x = blockX + threadIdx.x;
			y = blockY + threadIdx.y;
		}
		else {
			//~18 ms (but with many uncoalesced reads)
			smImageBlock[threadIdx.y+radius][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y, iWidth);

			if (threadIdx.x < radius) {
				if (x-radius >= 0)
					smImageBlock[threadIdx.y+radius][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y, iWidth);
				else
					smImageBlock[threadIdx.y+radius][threadIdx.x] = 0;

				if (threadIdx.y < radius) {
					if (y-radius >= 0 && x-radius >= 0)
						smImageBlock[threadIdx.y][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y-radius, iWidth);
					else
						smImageBlock[threadIdx.y][threadIdx.x] = 0;
				}
				else if (threadIdx.y + radius >= 16) {
					if (y+radius < iHeight && x-radius >= 0)
						smImageBlock[threadIdx.y+radius*2][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y+radius, iWidth);
					else
						smImageBlock[threadIdx.y+radius*2][threadIdx.x] = 0;
				}
			}
			else if (threadIdx.x + radius >= 16) {
				if (x+radius < iWidth)
					smImageBlock[threadIdx.y+radius][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y, iWidth);
				else
					smImageBlock[threadIdx.y+radius][threadIdx.x+radius*2] = 0;

				if (threadIdx.y < radius) {
					if (y-radius >= 0 && x+radius < iWidth)
						smImageBlock[threadIdx.y][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y-radius, iWidth);
					else
						smImageBlock[threadIdx.y][threadIdx.x+radius*2] = 0;
				}
				else if (threadIdx.y + radius >= 16) {
					if (y+radius < iHeight && x+radius < iWidth)
						smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y+radius, iWidth);
					else
						smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius*2] = 0;
				}
			}

			if (threadIdx.y < radius) {
				if (y-radius >= 0)
					smImageBlock[threadIdx.y][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y-radius, iWidth);
				else
					smImageBlock[threadIdx.y][threadIdx.x+radius] = 0;
			}
			else if (threadIdx.y + radius >= 16) {
				if (y+radius < iHeight)
					smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y+radius, iWidth);
				else
					smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius] = 0;
			}
		}


		data = 0;

		__syncthreads();

		for (j = threadIdx.y; j < threadIdx.y+blockSize; j++) { // 36 FLOP
			/*
			for (x = threadIdx.x; x < threadIdx.x+blockSize; x++) {
				data += smImageBlock[y][x];
			}
			*/ 
			data += sumValues<blockSize>(&smImageBlock[j][threadIdx.x]); // about 3ms faster than a loop
		}

		setArrayValue(d_imageOut, data, x, y, iWidth);
	}
}

template <class T>
__global__ void setArray(T* arr, T value, int width, int height) {
	int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;

	if (x < width && y < height) {
		arr[y*width+x] = value;
	}
}

template <bool RL>
__global__ void refineSubPixel(float *d_DMI, int *d_Cost, unsigned int *d_DSI, int disp_start, int disp_stop, int disp_step, int iWidth, int iHeight) {
	const int disparities = (disp_stop - disp_start + 1) / disp_step;
	int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	int minDisp, minCost, curCost;
	int predecessor, successor;
	int d;
	int offset1, offset2;
	float subPixelDisp;
	int disp_cut;

	if (x < iWidth && y < iHeight) {
		minDisp = -1;
		minCost = ~(1<<31);

		offset1 = IMUL(iWidth, iHeight);
		offset2 = IMUL(y, iWidth);

		if (RL) {
			for (d = 0; d < disparities; d++) {
				disp_cut = disp_start+disp_step*d;

				if (x < iWidth-disp_cut) {
					// load w/o texture, because loading coalesces
					curCost = d_DSI[d*offset1 + offset2 + x];
					//curCost = tex1Dfetch(texDSI1d, d*offset1 + offset2 + x);

					if (curCost <= minCost) {
						minDisp = d;
						minCost = curCost;
					}
				}
			}
		}
		else {
			for (d = 0; d < disparities; d++) {
				disp_cut = disp_start+disp_step*d;

				if (x >= disp_cut) {
					//curCost = d_DSI[d*offset1 + offset2 + x-disp_cut];
					curCost = tex1Dfetch(texDSI1d, d*offset1 + offset2 + x-disp_cut);

					if (curCost <= minCost) {
						minDisp = d;
						minCost = curCost;
					}
				}
			}
		}

		if (minDisp > 0 && minDisp < disp_stop) {
			if (RL) {
				//predecessor = getArrayValue(d_DSI, x, y, minDisp-1, iWidth, iHeight);
				//successor = getArrayValue(d_DSI, x, y, minDisp+1, iWidth, iHeight);
				predecessor = tex1Dfetch(texDSI1d, (minDisp-1)*offset1 + offset2 + x);
				successor = tex1Dfetch(texDSI1d, (minDisp+1)*offset1 + offset2 + x);
			}
			else {
				//predecessor = getArrayValue(d_DSI, x-disp_start-(minDisp-1)*disp_step, y, minDisp-1, iWidth, iHeight);
				//successor = getArrayValue(d_DSI, x-disp_start-(minDisp+1)*disp_step, y, minDisp+1, iWidth, iHeight);
				predecessor = tex1Dfetch(texDSI1d, (minDisp-1)*offset1 + offset2 + x-disp_start-(minDisp-1)*disp_step);
				successor = tex1Dfetch(texDSI1d, (minDisp+1)*offset1 + offset2 + x-disp_start-(minDisp+1)*disp_step);
			}

			if (2*minCost-predecessor-successor == 0)
				subPixelDisp = (minDisp * disp_step + disp_start);
			else {
				subPixelDisp = (minDisp * disp_step + disp_start) + ((successor-predecessor) / (float)(2*(2*minCost-predecessor-successor)));

				if (subPixelDisp > disp_stop)
					subPixelDisp = disp_stop;
				else if (subPixelDisp < disp_start)
					subPixelDisp = disp_start;
			}
		}
		else {
			subPixelDisp = (minDisp * disp_step + disp_start);
		}
      
		setArrayValue(d_DMI, subPixelDisp, x, y, iWidth);
		setArrayValue(d_Cost, minCost, x, y, iWidth);
	}
}


__global__ void compareDispsLeft(float *d_DMI, int *d_Costs, float *d_DMI_LR, float *d_DMI_RL, int *d_CostsLR, int *d_CostsRL, float max_disp_diff, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;

	float a, b, diff;


	if (x < iWidth && y < iHeight) {
		a = getArrayValue(d_DMI_LR, x, y, iWidth);
		if (a > x || x-a >= iWidth) b = 0; // for debuging only
		else b = getArrayValue(d_DMI_RL, f2i(x-a), y, iWidth);
		diff = abs(a - b);

		if (diff == a) {
			setArrayValue(d_DMI, a, x, y, iWidth);
			setArrayValue(d_Costs, getArrayValue(d_CostsLR, x, y, iWidth), x, y, iWidth);
		}
		else if (diff == b) {
			if (x+b < iWidth) {
				setArrayValue(d_DMI, b, f2i(x+b), y, iWidth);
				setArrayValue(d_Costs, getArrayValue(d_CostsRL, f2i(x+b), y, iWidth), f2i(x+b), y, iWidth);
			}
		}
		else if (diff <= max_disp_diff) {
			setArrayValue(d_DMI, (a+b)/2, x, y, iWidth);
			setArrayValue(d_Costs, getArrayValue(d_CostsLR, x, y, iWidth), x, y, iWidth);
		}
		else {
			setArrayValue(d_DMI, 0.0f, x, y, iWidth);
			setArrayValue(d_Costs, 0, x, y, iWidth);
		}
	}
}

__global__ void roundDisparities(float *d_DMI, int resolution, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		value *= resolution;
		value += 0.5f;
		value = (float)((int)value) / resolution;

		setArrayValue(d_DMI, value, x, y, iWidth);
	}
}

__global__ void scaleDisparities(float *d_DMI, int scale, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		value *= scale;

		setArrayValue(d_DMI, value, x, y, iWidth);
	}
}

__global__ void roundAndScaleDisparities(float *d_DMI, int resolution, int scale, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		// round
		value *= resolution;
		value += 0.5f;
		value = (float)((int)value) / resolution;

		// scale
		value *= scale;

		setArrayValue(d_DMI, value, x, y, iWidth);
	}
}


// takes about 1.84 ms for a 16x16 census
extern "C" void gpuCensusTransform() {

	hipStreamSynchronize( streamImageLeft[g_streamNr] ) ;
	hipStreamSynchronize( streamImageRight[g_streamNr] ) ;

	START_TIMER;

	if (g_sparse) {
		dim3 grid(iDivUp(g_Width, CENSUS_SPARSE_TILE_W), iDivUp(g_Height, CENSUS_SPARSE_TILE_H));
		dim3 block(16, 16);

		// left
		censusTransformSparse<true> <<<grid, block>>>(d_censusLeft, g_Width, g_Height);

		// right
		censusTransformSparse<false> <<<grid, block>>>(d_censusRight, g_Width, g_Height);

//		printf("  Bandwidth: %.1f GB/s\n", 2*(iDivUp(g_Width, CENSUS_SPARSE_TILE_W)*iDivUp(g_Height, CENSUS_SPARSE_TILE_H)*CENSUS_SPARSE_SMEM_W*CENSUS_SMEM_H*sizeof(unsigned int)+g_Width*g_Height*8)/(gpuTime*1000000));
//		printf("  GFLOPS:  : %.1f\n", 2*3000*16*16*iDivUp(g_Width, 64)*iDivUp(g_Height, 64)/(gpuTime*1000000));
	}
	else {
		dim3 grid(iDivUp(g_Width, CENSUS_TILE_W), iDivUp(g_Height, CENSUS_TILE_H));
		dim3 block(8, 24);

		// left
		censusTransform<true> <<<grid, block>>>(d_censusLeft, g_Width, g_Height);

		// right
		censusTransform<false> <<<grid, block>>>(d_censusRight, g_Width, g_Height);
	}

	STOP_TIMER(g_CensusTiming[eCensusTransform]);
}

extern "C" void gpuCalcDSI() {
	unsigned int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;
	int sharedMem;

	START_TIMER;

	// ((2 * (#threads + maxdisparity)) * nrOfInt) int values must be allocated
	if (g_sparse) {
		dim3 grid(iDivUp(g_Width, 128), g_Height);
		dim3 block(128, 1);

		sharedMem = 2 * (128 + disparities) * 2 * sizeof(unsigned int);

#if USE_FAST_AGGREGATION
		calcDSI<2> <<<grid, block, sharedMem>>>(d_DSI_Temp, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
#else
		calcDSI<2> <<<grid, block, sharedMem>>>(d_DSI, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
#endif
	}
	else {
		dim3 grid(iDivUp(g_Width, 128), g_Height);
		dim3 block(128, 1);

		sharedMem = 2 * (128 + disparities) * 8 * sizeof(unsigned int);

#if USE_FAST_AGGREGATION
		calcDSI<8> <<<grid, block, sharedMem>>>(d_DSI_Temp, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
#else
		calcDSI<8> <<<grid, block, sharedMem>>>(d_DSI, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
#endif	
	}

	STOP_TIMER(g_CensusTiming[eCalcDSI]);

//	printf("  Bandwidth: %.1f GB/s\n", (iDivUp(g_Width, 128)*g_Height*2*(128 + disparities)*2*sizeof(unsigned int)+2*(g_Width-g_disp_max)*g_Height*sizeof(unsigned int))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", (5*(128+disparities)*2*g_Height*iDivUp(g_Width, 128)+(17+43*disparities)*g_Width*g_Height)/(gpuTime*1000000));
}

extern "C" void gpuAggregateCosts() {
	int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;
	int d;

	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

#if USE_FAST_AGGREGATION
	for (d = 0; d < disparities; d++) {
		aggregateFilter<false, 5> <<<grid, block>>> (&d_DSI[g_Width*g_Height*d], &d_DSI_Temp[g_Width*g_Height*d], g_Width, g_Height);
	}
#else
	unsigned int *d_Temp;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Temp, g_Width * g_Height * sizeof(unsigned int)));

	for (d = 0; d < disparities; d++) {
		CUDA_SAFE_CALL( hipMemcpy(d_Temp, &d_DSI[g_Width*g_Height*d], g_Width*g_Height*sizeof(unsigned int), hipMemcpyDeviceToDevice) );
		aggregateFilter<false, 5> <<<grid, block>>> (&d_DSI[g_Width*g_Height*d], d_Temp, g_Width, g_Height);
	}

	CUDA_SAFE_CALL( hipFree(d_Temp) );
#endif

	STOP_TIMER(g_CensusTiming[eAggregateCosts]);

//	printf("  Bandwidth: %.1f GB/s\n", disparities*(g_Width*g_Height*sizeof(int) + iDivUp(g_Width, 16)*iDivUp(g_Height, 16)*21*21*sizeof(int))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", disparities*60*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuRefineSubPixel() {
	/*
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 8));
	dim3 block(16, 8);
	*/
	dim3 grid(iDivUp(g_Width, 128), iDivUp(g_Height, 1));
	dim3 block(128, 1);

	START_TIMER;

	refineSubPixel<true> <<<grid, block>>>(d_DMI_RL, d_Costs_RL, d_DSI, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);

	//printf("  Bandwidth: %.1f GB/s\n", ((disparities+2)*g_Width*g_Height*sizeof(int))/(gpuTime*1000000));
	//printf("  GFLOPS:  : %.1f\n", ((disparities*7+35)*g_Width*g_Height) /(gpuTime*1000000) );

	refineSubPixel<false> <<<grid, block>>>(d_DMI_LR, d_Costs_LR, d_DSI, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eRefineSubPixel]);

	//printf("  Bandwidth: %.1f GB/s\n", ((disparities+2)*g_Width*g_Height*sizeof(int))/(gpuTime*1000000));
	//printf("  GFLOPS:  : %.1f\n", ((disparities*7+43)*g_Width*g_Height) /(gpuTime*1000000) );
}

extern "C" void gpuCompareDisps() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	compareDispsLeft<<<grid, block>>>(d_DMI, d_Costs, d_DMI_LR, d_DMI_RL, d_Costs_LR, d_Costs_RL, 1.0, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eCompareDisps]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuRoundAndScaleDisparities() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	roundAndScaleDisparities<<<grid, block>>>(d_DMI, 10, 4, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eRoundAndScaleDisparities]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 12*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuCensusImageSetup(unsigned int w, unsigned int h, unsigned int disp_min, unsigned int disp_max, unsigned int disp_step, bool sparse, unsigned int blockSize) {
	unsigned int disparities = (disp_max + 1 - disp_min) / disp_step;

	g_Width = w;
	g_Height = h;

	g_disp_min = disp_min;
	g_disp_max = disp_max;
	g_disp_step = disp_step;
	g_sparse = sparse;
	g_blockSize = blockSize;

	for (int i=0; i<2; i++) {
		CUDA_SAFE_CALL( hipStreamCreate(&streamImageLeft[i]) ); 
		CUDA_SAFE_CALL( hipStreamCreate(&streamImageRight[i]) ); 
		CUDA_SAFE_CALL( hipStreamCreate(&streamDM[i]) ); 

		CUDA_SAFE_CALL( hipHostMalloc((void**)&h_left[i],  g_Width*g_Height*sizeof(unsigned char)) );
		CUDA_SAFE_CALL( hipHostMalloc((void**)&h_right[i], g_Width*g_Height*sizeof(unsigned char)) );
		CUDA_SAFE_CALL( hipHostMalloc((void**)&h_dm[i],    g_Width*g_Height*sizeof(unsigned char)) );
	}

	hipChannelFormatDesc channelDescChar = hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc channelDescInt = hipCreateChannelDesc<unsigned int>();

	CUDA_SAFE_CALL(hipMallocArray(&cuImageLeft, &channelDescChar, g_Width, g_Height));
	CUDA_SAFE_CALL(hipMallocArray(&cuImageRight, &channelDescChar, g_Width, g_Height));
	CUDA_SAFE_CALL(hipBindTextureToArray(texImageLeft,  cuImageLeft, channelDescChar));
	CUDA_SAFE_CALL(hipBindTextureToArray(texImageRight, cuImageRight, channelDescChar));

	if (sparse) {
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusLeft, g_Width*g_Height*8));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusRight, g_Width*g_Height*8));
	}
	else {
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusLeft, g_Width*g_Height*256/8));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusRight, g_Width*g_Height*256/8));
	}

	CUDA_SAFE_CALL(hipMalloc((void**)&d_DSI, disparities * g_Width * g_Height * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipBindTexture(0, texDSI1d, d_DSI, disparities * g_Width * g_Height * sizeof(unsigned int)));

#if USE_FAST_AGGREGATION
	CUDA_SAFE_CALL(hipMalloc((void**)&d_DSI_Temp, disparities * g_Width * g_Height * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipBindTexture(0, texDSITemp1d, d_DSI_Temp, disparities * g_Width * g_Height * sizeof(unsigned int)));
#endif

	//CUDA_SAFE_CALL(hipMalloc((void**)&d_integralImage, ((g_Width-disp_max) + 2 * g_blockSize) * (g_Height + 2 * g_blockSize) * sizeof(unsigned int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI, g_Width * g_Height * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI2, g_Width * g_Height * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI_RL, g_Width * g_Height * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI_LR, g_Width * g_Height * sizeof(float)));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_Costs, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Costs_RL, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Costs_LR, g_Width * g_Height * sizeof(int)));
}

extern "C" void gpuCensusImageCleanup() {
	CUDA_SAFE_CALL( hipUnbindTexture(texImageLeft) );
	CUDA_SAFE_CALL( hipUnbindTexture(texImageRight) );

	for (int i=0; i<2; i++) {
		CUDA_SAFE_CALL( hipStreamDestroy(streamImageLeft[i]) ); 
		CUDA_SAFE_CALL( hipStreamDestroy(streamImageRight[i]) ); 
		CUDA_SAFE_CALL( hipStreamDestroy(streamDM[i]) ); 

		CUDA_SAFE_CALL( hipHostFree(h_left[i]) );
		CUDA_SAFE_CALL( hipHostFree(h_right[i]) );
		CUDA_SAFE_CALL( hipHostFree(h_dm[i]) );
	}

	CUDA_SAFE_CALL( hipFreeArray(cuImageLeft) );
	CUDA_SAFE_CALL( hipFreeArray(cuImageRight) );

    CUDA_SAFE_CALL( hipFree(d_censusLeft) );
    CUDA_SAFE_CALL( hipFree(d_censusRight) );
    CUDA_SAFE_CALL( hipFree(d_DSI) );
#if USE_FAST_AGGREGATION
    CUDA_SAFE_CALL( hipFree(d_DSI_Temp) );
#endif
    CUDA_SAFE_CALL( hipFree(d_DMI) );
    CUDA_SAFE_CALL( hipFree(d_DMI2) );
    CUDA_SAFE_CALL( hipFree(d_DMI_LR) );
    CUDA_SAFE_CALL( hipFree(d_DMI_RL) );
    CUDA_SAFE_CALL( hipFree(d_Costs) );
    CUDA_SAFE_CALL( hipFree(d_Costs_LR) );
    CUDA_SAFE_CALL( hipFree(d_Costs_RL) );
}

extern "C" unsigned char* gpuGetLeftImageBuffer(int nr) {
	if (nr == 0 || nr == 1)
		return h_left[nr];
	else
		return NULL;
}

extern "C" unsigned char* gpuGetRightImageBuffer(int nr) {
	if (nr == 0 || nr == 1)
		return h_right[nr];
	else
		return NULL;
}

extern "C" void gpuCensusSetAsyncImageNr(int nr) {
	g_streamNr = nr;
}

extern "C" void gpuCensusLoadImages(int nr) {
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageLeft, 0, 0, h_left[nr], DATA_SIZE, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageRight, 0, 0, h_right[nr], DATA_SIZE, hipMemcpyHostToDevice) );
}

extern "C" void gpuCensusSetImages(unsigned char *left, unsigned char *right) {
	unsigned int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;

	//CUDA_SAFE_CALL(hipMemset(d_integralImage, 0, ((g_Width-disp_max) + 2 * g_blockSize) * (g_Height + 2 * g_blockSize) * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemset(d_DSI, 0, disparities * g_Width * g_Height * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemset(d_Costs, 0, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_Costs_RL, 0, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_Costs_LR, 0, g_Width * g_Height * sizeof(int)));

	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	setArray<<<grid, block>>>(d_DMI, 0.f, g_Width, g_Height);
	setArray<<<grid, block>>>(d_DMI_RL, -1.f, g_Width, g_Height);
	setArray<<<grid, block>>>(d_DMI_LR, -1.f, g_Width, g_Height);
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageLeft, 0, 0, left, DATA_SIZE, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageRight, 0, 0, right, DATA_SIZE, hipMemcpyHostToDevice) );
}

extern "C" void gpuGetDisparityMap(float *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_DMI, g_Width*g_Height*sizeof(float), hipMemcpyDeviceToHost) );
}

extern "C" void debugGetCensusLeft(int *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_censusLeft, g_Width*g_Height*sizeof(int)*2, hipMemcpyDeviceToHost) );
}
extern "C" void debugGetCensusRight(int *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_censusRight, g_Width*g_Height*sizeof(int)*8, hipMemcpyDeviceToHost) );
}
extern "C" void debugGetDSI(int *h_data, int d) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, &d_DSI[g_Width*g_Height*d], g_Width*g_Height*sizeof(int), hipMemcpyDeviceToHost) );
}
extern "C" void debugGetDMI_LR(float *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_DMI_LR, g_Width*g_Height*sizeof(float), hipMemcpyDeviceToHost) );
}
extern "C" void debugGetDMI_RL(float *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_DMI_RL, g_Width*g_Height*sizeof(float), hipMemcpyDeviceToHost) );
}
