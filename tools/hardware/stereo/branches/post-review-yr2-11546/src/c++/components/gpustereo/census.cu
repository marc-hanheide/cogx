#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "align.h"

#include "census.h"



unsigned int g_Width, g_Height;
unsigned int g_disp_min, g_disp_max, g_disp_step;
unsigned int g_blockSize;
int g_dmScale;
bool g_sparse;
int g_streamNr;

// CUDA arrays
hipArray *cuImageLeft, *cuImageRight;

// CUDA streams
hipStream_t streamImageLeft[2], streamImageRight[2], streamDM[2];

// Textures
texture<unsigned char, 2, hipReadModeElementType> texImageLeft;
texture<unsigned char, 2, hipReadModeElementType> texImageRight;
texture<unsigned int, 1, hipReadModeElementType> texDSI1d;
texture<unsigned int, 1, hipReadModeElementType> texDSITemp1d;

// Host-side page-locked memory
unsigned char *h_left[2], *h_right[2], *h_dm[2];

// device data pointer
unsigned int *d_DSI;
unsigned int *d_DSI_Temp;
//unsigned int *d_DSI_Temp2;
unsigned int *d_censusLeft, *d_censusRight;
int *d_Costs, *d_Costs_LR, *d_Costs_RL;
unsigned int *d_Confidence, *d_Confidence_LR, *d_Confidence_RL;
float *d_DMI, *d_DMI_LR, *d_DMI_RL;
unsigned int *d_Texture;
unsigned int *d_DepthMap;



float g_CensusTiming[10];

#define AGGFILTERWIDTH 3

#define PRINT_BANDWIDTH 0
#define USE_FAST_AGGREGATION 1

// 24 bit multiplication takes only 4 clock cycles compared to 16 clock 
// cycles for the normal multiplication
#define USE_24BIT_MULTIPLICATION 1

#if USE_24BIT_MULTIPLICATION
	#define IMUL(a, b) __mul24(a, b)
#else
	#define IMUL(a, b) ((a) * (b))
#endif

#define DATA_SIZE (g_Width*g_Height*sizeof(unsigned char))

#define CENSUS_RADIUS 8
#define CENSUS_TILE_W 64
#define CENSUS_TILE_H 64
#define CENSUS_SMEM_W (2 * CENSUS_RADIUS + CENSUS_TILE_W)
#define CENSUS_SMEM_H (2 * CENSUS_RADIUS + CENSUS_TILE_H)

#define CENSUS_SPARSE_RADIUS 8
#define CENSUS_SPARSE_TILE_W 64
#define CENSUS_SPARSE_TILE_H 32
#define CENSUS_SPARSE_SMEM_W (2 * CENSUS_SPARSE_RADIUS + CENSUS_SPARSE_TILE_W)
#define CENSUS_SPARSE_SMEM_H (2 * CENSUS_SPARSE_RADIUS + CENSUS_SPARSE_TILE_H)
#define CENSUS_SPARSE_THREAD_X (CENSUS_SPARSE_TILE_W / 16)
#define CENSUS_SPARSE_THREAD_Y (CENSUS_SPARSE_TILE_H / 16)

#define START_TIMER \
		unsigned int hTimer; \
		CUT_SAFE_CALL(cutCreateTimer(&hTimer)); \
		CUDA_SAFE_CALL( hipDeviceSynchronize() ); \
		CUT_SAFE_CALL( cutResetTimer(hTimer) ); \
		CUT_SAFE_CALL( cutStartTimer(hTimer) );

#define STOP_TIMER(gpuTime) \
		CUDA_SAFE_CALL( hipDeviceSynchronize() ); \
		CUT_SAFE_CALL(cutStopTimer(hTimer)); \
		gpuTime = cutGetTimerValue(hTimer);


void setCensusTiming(CensusStep s, float timeMs) {
	g_CensusTiming[s] = timeMs;
}

extern "C" float getCensusTiming(CensusStep s) {
	return g_CensusTiming[s];
}

extern "C" unsigned int getCensusFLOP(CensusStep s) {
	int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;

	switch (s) {
		case eAggregateCosts:
			return disparities*60*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		case eCalcDSI:
			return (5*(128+disparities)*2*g_Height*iDivUp(g_Width, 128)+(17+43*disparities)*g_Width*g_Height);
		
		case eCensusTransform:
			return 2*3000*16*16*iDivUp(g_Width, 64)*iDivUp(g_Height, 64);
		
		case eCompareDisps:
			return 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		case eRefineSubPixel:
			return ((disparities*7+35)*g_Width*g_Height) + ((disparities*7+43)*g_Width*g_Height);
		
		case eRoundAndScaleDisparities:
			return 12*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		case eThresholdConfidence:
			return 8*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		case eThresholdTexture:
			return 8*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		case eCalcTextureMap:
			return 151*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);

		case eCalcDepthMap:
			return 11*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16);
		
		default:
			return 0;
	}
}

extern "C" unsigned int getCensusMemory(CensusStep s) {
	int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;

	switch (s) {
		case eAggregateCosts:
			return disparities*(g_Width*g_Height*sizeof(int) + iDivUp(g_Width, 16)*iDivUp(g_Height, 16)*21*21*sizeof(int));
		
		case eCalcDSI:
			return (iDivUp(g_Width, 128)*g_Height*2*(128 + disparities)*2*sizeof(unsigned int)+2*(g_Width-g_disp_max)*g_Height*sizeof(unsigned int));
		
		case eCensusTransform:
			return 2*(iDivUp(g_Width, CENSUS_SPARSE_TILE_W)*iDivUp(g_Height, CENSUS_SPARSE_TILE_H)*CENSUS_SPARSE_SMEM_W*CENSUS_SMEM_H*sizeof(unsigned int)+g_Width*g_Height*8);
		
		case eCompareDisps:
			return 2*g_Width*g_Height*sizeof(float);
		
		case eRefineSubPixel:
			return 2*(disparities+2)*g_Width*g_Height*sizeof(int);
		
		case eRoundAndScaleDisparities:
			return 2*g_Width*g_Height*sizeof(float);
		
		case eThresholdConfidence:
			return 2*g_Width*g_Height*sizeof(int);
		
		case eThresholdTexture:
			return 2*g_Width*g_Height*sizeof(int);

		case eCalcTextureMap:
			return 7*4*g_Width*g_Height*sizeof(char) + g_Width*g_Height*sizeof(int);

		case eCalcDepthMap:
			return 2*g_Width*g_Height*sizeof(unsigned int);

		default:
			return 0;
	}
}


inline
__device__ int f2i(float f) {

	return f < 0 ? f - .5 : f + .5; 

}

template <typename T>
inline
__device__ T getArrayValue(const T *arr, int x, int y, int width) {

	return arr[IMUL(y, width) + x];

}

template <typename T>
inline
__device__ T getArrayValue(const T *arr, int x, int y, int z, int width, int height) {

	return arr[z * IMUL(width, height) + IMUL(y, width) + x];

}

template <typename T>
inline
__device__ void setArrayValue(T *arr, T value, int x, int y, int width) {

	arr[IMUL(y, width) + x] = value;

}

template <typename T>
inline
__device__ void setArrayValue(T *arr, T value, int x, int y, int z, int width, int height) {

	arr[z * IMUL(width, height) + IMUL(y, width) + x] = value;

}

template <int c>
__device__ unsigned int sumValuesSparse(unsigned char* arr) {
	return sumValuesSparse<c-2>(arr) + arr[c-2];
}

template <>
__device__ unsigned int sumValuesSparse<0>(unsigned char* arr) {
	return 0;
}

template <int c>
__device__ unsigned int squareSumValuesSparse(unsigned char* arr) {
	return squareSumValuesSparse<c-2>(arr) + (arr[c-2]*arr[c-2]);
}

template <>
__device__ unsigned int squareSumValuesSparse<0>(unsigned char* arr) {
	return 0;
}

inline
__device__ unsigned char censusRowSparse(unsigned char ref, unsigned char *row) {
	unsigned char c = 0;

	if (ref > row[ 0]) c |= (1 << 7);
	if (ref > row[ 2]) c |= (1 << 6);
	if (ref > row[ 4]) c |= (1 << 5);
	if (ref > row[ 6]) c |= (1 << 4);
	if (ref > row[ 8]) c |= (1 << 3);
	if (ref > row[10]) c |= (1 << 2);
	if (ref > row[12]) c |= (1 << 1);
	if (ref > row[14]) c |= (1 << 0);

	return c;
}


__global__ void calcTextureMapSM(unsigned int *d_texture, int iWidth, int iHeight) {
	const unsigned int blockX = IMUL(blockDim.x, blockIdx.x);
	const unsigned int blockY = IMUL(blockDim.y, blockIdx.y);
	int i, j;

	__shared__ unsigned char smTileData[16+6][16+12+1];
/*
	for (j=0; j<16+6; j+=16)
		for (i=0; i<16+12; i+=16)
			smTileData[threadIdx.y+j][threadIdx.x+i] = tex2D(texImageLeft, blockX+i-6+threadIdx.x, blockY+j-3+threadIdx.y);
*/

	for (j = threadIdx.y; j < 16+6; j+=blockDim.y) 
		for (i = threadIdx.x; i < 16+12; i+=blockDim.x)
			smTileData[j][i] = tex2D(texImageLeft, blockX+i-6, blockY+j-3);

	__syncthreads();

	unsigned int x = blockX + threadIdx.x;
	unsigned int y = blockY + threadIdx.y;

	if (x < iWidth && y < iHeight) {
		/*
		int sum = sumValuesSparse<12>(&smTileData[threadIdx.y+3-3][threadIdx.x+6-6]) + \
				  sumValuesSparse<12>(&smTileData[threadIdx.y+3-1][threadIdx.x+6-6]) + \
				  sumValuesSparse<12>(&smTileData[threadIdx.y+3+1][threadIdx.x+6-6]) + \
				  sumValuesSparse<12>(&smTileData[threadIdx.y+3+3][threadIdx.x+6-6]);
		int sumsqr = squareSumValuesSparse<12>(&smTileData[threadIdx.y+3-3][threadIdx.x+6-6]) + \
					 squareSumValuesSparse<12>(&smTileData[threadIdx.y+3-1][threadIdx.x+6-6]) + \
					 squareSumValuesSparse<12>(&smTileData[threadIdx.y+3+1][threadIdx.x+6-6]) + \
					 squareSumValuesSparse<12>(&smTileData[threadIdx.y+3+3][threadIdx.x+6-6]);
					 */

		int sum = 0;
		int sumsqr = 0;
		unsigned char pixel;

		for (j = -3; j<=3; j+=2) {
			for (i = -6; i <= 6; i+=2) {
				pixel = smTileData[threadIdx.y+3+j][threadIdx.x+6+i];

				sum += pixel;
				sumsqr += pixel*pixel;
			}
		}

		float nr = 13*7/4.;
		float tex = (sum/nr);
		tex *= tex;
		tex = abs(sumsqr/nr-tex);
		unsigned int itex = (unsigned int)tex;
		itex /= 8;
		if (itex > 255) itex = 255;

		setArrayValue(d_texture, itex, x, y, iWidth);
	}
}

__global__ void calcTextureMap(unsigned int *d_texture, int iWidth, int iHeight) {
	unsigned int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	unsigned int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	int i, j;
	unsigned char pixel;
	int sum = 0;
	int sumsqr = 0;

	if (x < iWidth && y < iHeight) {

		for (j = -3; j<=3; j+=2) {
			for (i = -6; i <= 6; i+=2) {
				pixel = tex2D(texImageLeft, x+i, y+j);

				sum += pixel;
				sumsqr += pixel*pixel;
			}
		}

	/*
		int sum = sumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS-3][threadIdx.x+CENSUS_SPARSE_RADIUS-6]) + \
				  sumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS-1][threadIdx.x+CENSUS_SPARSE_RADIUS-6]) + \
				  sumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS+1][threadIdx.x+CENSUS_SPARSE_RADIUS-6]) + \
				  sumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS+3][threadIdx.x+CENSUS_SPARSE_RADIUS-6]);
		int sumsqr = squareSumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS-3][threadIdx.x+CENSUS_SPARSE_RADIUS-6]) + \
					 squareSumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS-1][threadIdx.x+CENSUS_SPARSE_RADIUS-6]) + \
					 squareSumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS+1][threadIdx.x+CENSUS_SPARSE_RADIUS-6]) + \
					 squareSumValuesSparse<12>(&smTileData[threadIdx.y+CENSUS_SPARSE_RADIUS+3][threadIdx.x+CENSUS_SPARSE_RADIUS-6]);
	*/
		unsigned int tex = abs(sumsqr - (sum*sum) / (7*4));
		if (tex > 255) tex = 255;

		setArrayValue(d_texture, tex, x, y, iWidth);
	}
}

template <bool left>
__global__ void censusTransformSparseTex(unsigned int *d_census, int iWidth, int iHeight) {
	const int x = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	const int y = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
	int i, j;
	unsigned char p1;
	unsigned char p2;
	int2 c;

	if (x < iWidth && y < iHeight) {
		c.x = 0;
		c.y = 0;

		if (left) p1 = tex2D(texImageLeft, x, y);
		else      p1 = tex2D(texImageRight, x, y);

		for (j = -7; j <= -1; j+=2) {
			for (i = -7; i <= 7; i+=2) {
				if (left) p2 = tex2D(texImageLeft, x+i, y+j);
				else      p2 = tex2D(texImageRight, x+i, y+j);

				c.x <<= 1;
				if (p1 > p2) c.x |= 1;
			}
		}
		for (j = 1; j <= 7; j+=2) {
			for (i = -7; i <= 7; i+=2) {
				if (left) p2 = tex2D(texImageLeft, x+i, y+j);
				else      p2 = tex2D(texImageRight, x+i, y+j);

				c.y <<= 1;
				if (p1 > p2) c.y |= 1;
			}
		}

		__syncthreads();

		int idx = IMUL(y, iWidth) + x;
		((int2*)d_census)[idx] = c;
	}
}

template <bool left>
__global__ void censusTransformSparse(unsigned int *d_census, int iWidth, int iHeight) {
	const int blockX = IMUL(blockIdx.x, CENSUS_SPARSE_TILE_W);
	const int blockY = IMUL(blockIdx.y, CENSUS_SPARSE_TILE_H);
	int x, y;

	__shared__ unsigned char smTileData[CENSUS_SPARSE_SMEM_H][CENSUS_SPARSE_SMEM_W];

	for (y = threadIdx.y; y < CENSUS_SPARSE_SMEM_H; y+=blockDim.y) { // 130 FLOP
		for (x = threadIdx.x; x < CENSUS_SPARSE_SMEM_W; x+=blockDim.x) {
			if (left)
				smTileData[y][x] = tex2D(texImageLeft, blockX+x-CENSUS_SPARSE_RADIUS, blockY+y-CENSUS_SPARSE_RADIUS);
			else
				smTileData[y][x] = tex2D(texImageRight, blockX+x-CENSUS_SPARSE_RADIUS, blockY+y-CENSUS_SPARSE_RADIUS);
		}
	}


	// assure that shared memory is completely loaded
	__syncthreads();


	// start calculation
	int2 c;
	unsigned char p1;
	int xref, yref;
	int idx = 0;

	//for (yref = threadIdx.y * 4; yref < threadIdx.y * 4 + 4 && yref < CENSUS_SPARSE_TILE_H && blockY+yref < iHeight; yref++) { // 2868 FLOP
	for (yref = threadIdx.y * CENSUS_SPARSE_THREAD_Y; yref < threadIdx.y * CENSUS_SPARSE_THREAD_Y + CENSUS_SPARSE_THREAD_Y && yref < CENSUS_SPARSE_TILE_H && blockY+yref < iHeight; yref++) { // 2868 FLOP
		for (xref = threadIdx.x * CENSUS_SPARSE_THREAD_X; xref < threadIdx.x * CENSUS_SPARSE_THREAD_X + CENSUS_SPARSE_THREAD_X && xref < CENSUS_SPARSE_TILE_W && blockX+xref < iWidth; xref++) {
			p1 = smTileData[yref+CENSUS_SPARSE_RADIUS][xref+CENSUS_SPARSE_RADIUS];
			idx = (IMUL((blockY+yref), iWidth)+(blockX+xref));

			c.x = (censusRowSparse(p1, &smTileData[yref-7+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) << 24) | \
				  (censusRowSparse(p1, &smTileData[yref-5+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) << 16) | \
				  (censusRowSparse(p1, &smTileData[yref-3+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) <<  8) | \
				  (censusRowSparse(p1, &smTileData[yref-1+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) <<  0);

			c.y = (censusRowSparse(p1, &smTileData[yref+1+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) << 24) | \
				  (censusRowSparse(p1, &smTileData[yref+3+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) << 16) | \
				  (censusRowSparse(p1, &smTileData[yref+5+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) <<  8) | \
				  (censusRowSparse(p1, &smTileData[yref+7+CENSUS_SPARSE_RADIUS][xref-7+CENSUS_SPARSE_RADIUS]) <<  0);

			((int2*)d_census)[idx] = c;
		}
	}
}


template <bool left>
__global__ void censusTransform(unsigned int *d_census, int iWidth, int iHeight) {

	const int blockX = IMUL(blockIdx.x, CENSUS_TILE_W);
	const int blockY = IMUL(blockIdx.y, CENSUS_TILE_H);
	int x, y;

	__shared__ unsigned char smTileData[CENSUS_SMEM_H][CENSUS_SMEM_W];

	for (y = threadIdx.y; y < CENSUS_SMEM_H; y+=blockDim.y) { // 130 FLOP
		for (x = threadIdx.x; x < CENSUS_SMEM_W; x+=blockDim.x) {
			if (left)
				smTileData[y][x] = tex2D(texImageLeft, blockX+x-CENSUS_RADIUS, blockY+y-CENSUS_RADIUS);
			else
				smTileData[y][x] = tex2D(texImageRight, blockX+x-CENSUS_RADIUS, blockY+y-CENSUS_RADIUS);
		}
	}


	// assure that shared memory is completely loaded
	__syncthreads();


	// start calculation
	unsigned int c;
	char p1;
	char p2;
	int xref, yref;
	int idx = 0;
	int i;

	for (yref = threadIdx.y * 4; yref < threadIdx.y * 4 + 4 && yref < CENSUS_SPARSE_TILE_H && blockY+yref < iHeight; yref++) { // 2868 FLOP
		for (xref = threadIdx.x * 4; xref < threadIdx.x * 4 + 4 && xref < CENSUS_SPARSE_TILE_W && blockX+xref < iWidth; xref++) {
			p1 = smTileData[yref+CENSUS_RADIUS][xref+CENSUS_RADIUS];
			idx = (IMUL((blockY+yref), iWidth)+(blockX+xref))*8;

			for (i = 0; i < 8; i++) {
				c = 0;

				for (y = -7+i*2; y <= -6+i*2; y++) {
					for (x = -7; x <= 8; x++) {
						p2 = smTileData[yref+y+CENSUS_RADIUS][xref+x+CENSUS_RADIUS];

						c <<= 1;

						if (p1 > p2)
							c |= 1;
					}
				}

				d_census[idx+i] = c;
			}
		}
	}
}

template <int nrOfInt>
inline
__device__ int calcHamming(unsigned int *a, unsigned int *b) {
	unsigned int exor;
	int result = 0, j;

	for (j = 0; j < nrOfInt; j++)
	{
		exor = a[j] ^ b[j];

		// counts the set bits
		exor = exor - ((exor >> 1) & 0x55555555);
		exor = (exor & 0x33333333) + ((exor >> 2) & 0x33333333);
		exor = ((exor + (exor >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;

		result += exor;
	}

	return result;
}

template <int nrOfInt>
inline
__device__ int calcHamming2(unsigned int *a, unsigned int *b) {
	unsigned int exor;
	int result = 0, j, i;

	for (j = 0; j < nrOfInt; j++)
	{
		exor = a[j] ^ b[j];

		for (i = 0; i < sizeof(unsigned int)*8; i++, exor >>= 1) {
			result += (exor & 0x1u);
		}
	}

	return result;
}

template <int blockSize, int nrOfInt>
__global__ void calcDSI(unsigned int *DSI, unsigned int *census_L, unsigned int *census_R, int disp_start, int disp_stop, int disp_step, int iWidth, int iHeight) {
	const int disparities = (disp_stop - disp_start + 1) / disp_step;
	//const int smWidth = (blockDim.x + disparities) * nrOfInt;
	const int smWidth = blockDim.x * nrOfInt;
	const int x = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	const int y = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
	const int blockX = IMUL(blockIdx.x, blockDim.x) * nrOfInt;
	int offset;
	int index;
	int i;

	// ((2 * (#threads + maxdisparity)) * nrOfInt) int values must be allocated
	extern __shared__ unsigned int smCensusData[];
	
	unsigned int *smCensusL = smCensusData;
	unsigned int *smCensusR = &smCensusData[smWidth+disp_stop*nrOfInt];

	offset = IMUL(y, iWidth) * nrOfInt + blockX;
	//for (i = threadIdx.x; i < smWidth && i + blockX < iWidth * nrOfInt; i += blockDim.x) { //5*(blockDim.x + disparities) * nrOfInt FLOPS
	//	smCensusL[i] = census_L[offset+i];
	//	smCensusR[i] = census_R[offset+i];
	//}

	for (i = threadIdx.x; i < smWidth && i + blockX < iWidth * nrOfInt; i += blockDim.x) { //5*(blockDim.x + disparities) * nrOfInt FLOPS
		smCensusL[i] = census_L[offset+i];
		smCensusR[i] = census_R[offset+i];
	}
	for (i = threadIdx.x + smWidth; i < smWidth + disp_stop * nrOfInt && i + blockX < iWidth * nrOfInt; i += blockDim.x) {
		smCensusL[i] = census_L[offset+i];
	}



	unsigned int cost;
	int offset2;
	
	__syncthreads();

	//Calculation
	if (x < iWidth && y < iHeight) {
		//offset  = IMUL(iWidth+blockSize, iHeight+blockSize); // 1 DSI page
		offset  = IMUL(iWidth, iHeight); // 1 DSI page
		//offset2 = IMUL(iWidth+blockSize, y+blockSize/2) + x+blockSize/2;   // index within DSI page
		offset2 = IMUL(iWidth, y) + x;   // index within DSI page

		for (i = disp_start; i < disparities + disp_start && x < iWidth-(disp_start+disp_step*i); i++) { //43 FLOP per cycle
			index = (i-disp_start) * offset + offset2;

			cost = calcHamming<nrOfInt>(&smCensusR[threadIdx.x * nrOfInt], &smCensusL[(threadIdx.x + i*disp_step + disp_start) * nrOfInt]);
			DSI[index] = cost;

			// synchronize to avoid serialization
			//__syncthreads();
		}
	}
}

/////////////////
// NVIDIA CODE
/////////////////


////////////////////////////////////////////////////////////////////////////////
// Kernel configuration
////////////////////////////////////////////////////////////////////////////////
#define KERNEL_RADIUS 2
#define      KERNEL_W (2 * KERNEL_RADIUS + 1)
__device__ __constant__ float d_Kernel[KERNEL_W];

// Assuming ROW_TILE_W, KERNEL_RADIUS_ALIGNED and dataW 
// are multiples of coalescing granularity size,
// all global memory operations are coalesced in convolutionRowGPU()
#define            ROW_TILE_W 128
#define KERNEL_RADIUS_ALIGNED 16

// Assuming COLUMN_TILE_W and dataW are multiples
// of coalescing granularity size, all global memory operations 
// are coalesced in convolutionColumnGPU()
#define COLUMN_TILE_W 16
#define COLUMN_TILE_H 32



////////////////////////////////////////////////////////////////////////////////
// Loop unrolling templates, needed for best performance
////////////////////////////////////////////////////////////////////////////////
template<int i> __device__ unsigned int convolutionRow(unsigned int *data){
    //return
    //    data[KERNEL_RADIUS - i] * d_Kernel[i]
    //    + convolutionRow<i - 1>(data);
	return
        data[KERNEL_RADIUS - i]
        + convolutionRow<i - 1>(data);
}

template<> __device__ unsigned int convolutionRow<-1>(unsigned int *data){
    return 0;
}

template<int i> __device__ unsigned int convolutionColumn(unsigned int *data){

    //return 
    //    data[(KERNEL_RADIUS - i) * COLUMN_TILE_W] * d_Kernel[i]
    //    + convolutionColumn<i - 1>(data);
		
    return 
        data[(KERNEL_RADIUS - i) * COLUMN_TILE_W]
        + convolutionColumn<i - 1>(data);
}

template<> __device__ unsigned int convolutionColumn<-1>(unsigned int *data){
    return 0;
}



////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowGPU(
    unsigned int *d_Result,
    unsigned int *d_Data,
    int dataW,
    int dataH
){
    //Data cache
    __shared__ unsigned int data[KERNEL_RADIUS + ROW_TILE_W + KERNEL_RADIUS];

    //Current tile and apron limits, relative to row start
    const int         tileStart = IMUL(blockIdx.x, ROW_TILE_W);
    const int           tileEnd = tileStart + ROW_TILE_W - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataW - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataW - 1);

    //Row start index in d_Data[]
    const int          rowStart = IMUL(blockIdx.y, dataW);

    //Aligned apron start. Assuming dataW and ROW_TILE_W are multiples 
    //of half-warp size, rowStart + apronStartAligned is also a 
    //multiple of half-warp size, thus having proper alignment 
    //for coalesced d_Data[] read.
    const int apronStartAligned = tileStart - KERNEL_RADIUS_ALIGNED;

    const int loadPos = apronStartAligned + threadIdx.x;
    //Set the entire data cache contents
    //Load global memory values, if indices are within the image borders,
    //or initialize with zeroes otherwise
    if(loadPos >= apronStart){
        const int smemPos = loadPos - apronStart;

        data[smemPos] = 
            ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ?
            d_Data[rowStart + loadPos] : 0;
    }


    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data,
    //loaded by another threads
    __syncthreads();
    const int writePos = tileStart + threadIdx.x;
    //Assuming dataW and ROW_TILE_W are multiples of half-warp size,
    //rowStart + tileStart is also a multiple of half-warp size,
    //thus having proper alignment for coalesced d_Result[] write.
    if(writePos <= tileEndClamped){
        const int smemPos = writePos - apronStart;
        unsigned int sum = 0;

        sum = convolutionRow<2 * KERNEL_RADIUS>(data + smemPos);

        d_Result[rowStart + writePos] = sum;
    }
}



////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnGPU(
    unsigned int *d_Result,
    unsigned int *d_Data,
    int dataW,
    int dataH,
    int smemStride,
    int gmemStride
){
    //Data cache
    __shared__ unsigned int data[COLUMN_TILE_W * (KERNEL_RADIUS + COLUMN_TILE_H + KERNEL_RADIUS)];

    //Current tile and apron limits, in rows
    const int         tileStart = IMUL(blockIdx.y, COLUMN_TILE_H);
    const int           tileEnd = tileStart + COLUMN_TILE_H - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataH - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataH - 1);

    //Current column index
    const int       columnStart = IMUL(blockIdx.x, COLUMN_TILE_W) + threadIdx.x;

    //Shared and global memory indices for current column
    int smemPos = IMUL(threadIdx.y, COLUMN_TILE_W) + threadIdx.x;
    int gmemPos = IMUL(apronStart + threadIdx.y, dataW) + columnStart;
    //Cycle through the entire data cache
    //Load global memory values, if indices are within the image borders,
    //or initialize with zero otherwise
    for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y){
        data[smemPos] = 
        ((y >= apronStartClamped) && (y <= apronEndClamped)) ? 
        d_Data[gmemPos] : 0;
        smemPos += smemStride;
        gmemPos += gmemStride;
    }

    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data, 
    //loaded by another threads
    __syncthreads();
    //Shared and global memory indices for current column
    smemPos = IMUL(threadIdx.y + KERNEL_RADIUS, COLUMN_TILE_W) + threadIdx.x;
    gmemPos = IMUL(tileStart + threadIdx.y , dataW) + columnStart;
    //Cycle through the tile body, clamped by image borders
    //Calculate and output the results
    for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y){
        unsigned int sum = 0;

        sum = convolutionColumn<2 * KERNEL_RADIUS>(data + smemPos);

        d_Result[gmemPos] = sum;
        smemPos += smemStride;
        gmemPos += gmemStride;
    }
}

/////////////////
/////////////////


template <int c>
__device__ unsigned int sumValues(unsigned int* arr) {
	return sumValues<c-1>(arr) + arr[c-1];
}

template <>
__device__ unsigned int sumValues<0>(unsigned int* arr) {
	return 0;
}

//template <bool computeCapability_1_2, int radius>
//__global__ void aggregateFilter(unsigned int *d_imageOut, unsigned int *d_imageIn, int iWidth, int iHeight) {
//	const int blockX = IMUL(blockDim.x, blockIdx.x);
//	const int blockY = IMUL(blockDim.y, blockIdx.y);
//	//const int radius = blockSize/2;
//	int x = blockX + threadIdx.x;
//	int y = blockY + threadIdx.y;
//	int i, j;
//	unsigned int data;
//
//	__shared__ unsigned int smImageBlock[16+radius*2][16+radius*2];
//
//
//	if (x < iWidth && y < iHeight) {
//		smImageBlock[threadIdx.y+radius][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y, iWidth);
//	}
//
//			if (threadIdx.x < radius) {
//				if (x-radius >= 0)
//					smImageBlock[threadIdx.y+radius][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y, iWidth);
//				else
//					smImageBlock[threadIdx.y+radius][threadIdx.x] = 0;
//
//				if (threadIdx.y < radius) {
//					if (y-radius >= 0 && x-radius >= 0)
//						smImageBlock[threadIdx.y][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y-radius, iWidth);
//					else
//						smImageBlock[threadIdx.y][threadIdx.x] = 0;
//				}
//				else if (threadIdx.y + radius >= 16) {
//					if (y+radius < iHeight && x-radius >= 0)
//						smImageBlock[threadIdx.y+radius*2][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y+radius, iWidth);
//					else
//						smImageBlock[threadIdx.y+radius*2][threadIdx.x] = 0;
//				}
//			}
//			else if (threadIdx.x + radius >= 16) {
//				if (x+radius < iWidth)
//					smImageBlock[threadIdx.y+radius][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y, iWidth);
//				else
//					smImageBlock[threadIdx.y+radius][threadIdx.x+radius*2] = 0;
//
//				if (threadIdx.y < radius) {
//					if (y-radius >= 0 && x+radius < iWidth)
//						smImageBlock[threadIdx.y][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y-radius, iWidth);
//					else
//						smImageBlock[threadIdx.y][threadIdx.x+radius*2] = 0;
//				}
//				else if (threadIdx.y + radius >= 16) {
//					if (y+radius < iHeight && x+radius < iWidth)
//						smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y+radius, iWidth);
//					else
//						smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius*2] = 0;
//				}
//			}
//
//			if (threadIdx.y < radius) {
//				if (y-radius >= 0)
//					smImageBlock[threadIdx.y][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y-radius, iWidth);
//				else
//					smImageBlock[threadIdx.y][threadIdx.x+radius] = 0;
//			}
//			else if (threadIdx.y + radius >= 16) {
//				if (y+radius < iHeight)
//					smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y+radius, iWidth);
//				else
//					smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius] = 0;
//			}
//
//		data = 0;
//
//		__syncthreads();
//		
//	if (x < iWidth && y < iHeight) {
//
//		//#pragma unroll
//		//for(j=0; j < radius*2+1; j++) {
//		//	#pragma unroll
//		//	for (i=0; i < radius*2+1; i++) {
//		//		data += smImageBlock[j+threadIdx.y][i+threadIdx.x];
//		//	}
//		//}
//
//		for (j = threadIdx.y; j < threadIdx.y+radius*2+1; j++) { // 36 FLOP
//			//for (x = threadIdx.x; x < threadIdx.x+blockSize; x++) {
//			//	data += smImageBlock[y][x];
//			//}
//			data += sumValues<radius*2+1>(&smImageBlock[j][threadIdx.x]); // about 3ms faster than a loop
//		}
//
//		setArrayValue(d_imageOut, data, x, y, iWidth);
//	}
//}

template <bool computeCapability_1_2, int radius>
__global__ void aggregateFilter(unsigned int *d_imageOut, unsigned int *d_imageIn, int iWidth, int iHeight) {
	const int blockX = IMUL(blockDim.x, blockIdx.x);
	const int blockY = IMUL(blockDim.y, blockIdx.y);
	//const int radius = blockSize/2;
	int x = blockX + threadIdx.x;
	int y = blockY + threadIdx.y;
	int i, j;
	unsigned int data;

	__shared__ unsigned int smImageBlock[16+radius*2][16+radius*2];

		if (computeCapability_1_2) {
			/* // ~40 ms
			for (y = threadIdx.y - radius; y < 16 + radius; y+=blockDim.y) {
				for (x = threadIdx.x - radius; x < 16 + radius; x+=blockDim.x) {
					if (x < 0 || y < 0 || x + blockX >= iWidth || y + blockY >= iHeight)
						data = 0;
					else
						data = getArrayValue(d_imageIn, blockX+x, blockY+y, iWidth);

					smImageBlock[y+radius][x+radius] = data;
				}
			}*/

			// Bad performance due to uncoalesced load, because -radius leads to 
			// misalignment on devices with Compute Capability < 1.2

			//~24 ms
			#pragma unroll
			for (j = 0; threadIdx.y + j < 16 + 2 * radius; j += 16) {
				y = blockY + threadIdx.y + j - radius;

				#pragma unroll
				for (i = 0; threadIdx.x + i < 16 + 2 * radius; i += 16) {
					x = blockX + threadIdx.x + i - radius;					

					if (x < 0 || y < 0 || x >= iWidth || y >= iHeight)
						data = 0;
					else
						data = getArrayValue(d_imageIn, x, y, iWidth);
						
					smImageBlock[threadIdx.y + j][threadIdx.x + i] = data;
				}
			}

			x = blockX + threadIdx.x;
			y = blockY + threadIdx.y;
		}


	if (x < iWidth && y < iHeight) {
		if (!computeCapability_1_2) {
			//~18 ms (but with many uncoalesced reads)
			smImageBlock[threadIdx.y+radius][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y, iWidth);

			if (threadIdx.x < radius) {
				if (x-radius >= 0)
					smImageBlock[threadIdx.y+radius][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y, iWidth);
				else
					smImageBlock[threadIdx.y+radius][threadIdx.x] = 0;

				if (threadIdx.y < radius) {
					if (y-radius >= 0 && x-radius >= 0)
						smImageBlock[threadIdx.y][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y-radius, iWidth);
					else
						smImageBlock[threadIdx.y][threadIdx.x] = 0;
				}
				else if (threadIdx.y + radius >= 16) {
					if (y+radius < iHeight && x-radius >= 0)
						smImageBlock[threadIdx.y+radius*2][threadIdx.x] = getArrayValue(d_imageIn, x-radius, y+radius, iWidth);
					else
						smImageBlock[threadIdx.y+radius*2][threadIdx.x] = 0;
				}
			}
			else if (threadIdx.x + radius >= 16) {
				if (x+radius < iWidth)
					smImageBlock[threadIdx.y+radius][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y, iWidth);
				else
					smImageBlock[threadIdx.y+radius][threadIdx.x+radius*2] = 0;

				if (threadIdx.y < radius) {
					if (y-radius >= 0 && x+radius < iWidth)
						smImageBlock[threadIdx.y][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y-radius, iWidth);
					else
						smImageBlock[threadIdx.y][threadIdx.x+radius*2] = 0;
				}
				else if (threadIdx.y + radius >= 16) {
					if (y+radius < iHeight && x+radius < iWidth)
						smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius*2] = getArrayValue(d_imageIn, x+radius, y+radius, iWidth);
					else
						smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius*2] = 0;
				}
			}

			if (threadIdx.y < radius) {
				if (y-radius >= 0)
					smImageBlock[threadIdx.y][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y-radius, iWidth);
				else
					smImageBlock[threadIdx.y][threadIdx.x+radius] = 0;
			}
			else if (threadIdx.y + radius >= 16) {
				if (y+radius < iHeight)
					smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius] = getArrayValue(d_imageIn, x, y+radius, iWidth);
				else
					smImageBlock[threadIdx.y+radius*2][threadIdx.x+radius] = 0;
			}
		}


		data = 0;

		__syncthreads();
		

		//#pragma unroll
		//for(j=0; j < radius*2+1; j++) {
		//	#pragma unroll
		//	for (i=0; i < radius*2+1; i++) {
		//		data += smImageBlock[j+threadIdx.y][i+threadIdx.x];
		//	}
		//}

		for (j = threadIdx.y; j < threadIdx.y+radius*2+1; j++) { // 36 FLOP
			//for (x = threadIdx.x; x < threadIdx.x+blockSize; x++) {
			//	data += smImageBlock[y][x];
			//}
			data += sumValues<radius*2+1>(&smImageBlock[j][threadIdx.x]); // about 3ms faster than a loop
		}

		setArrayValue(d_imageOut, data, x, y, iWidth);
	}
}


template <typename T>
__global__ void setArray(T* arr, T value, int width, int height) {
	int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;

	if (x < width && y < height) {
		arr[y*width+x] = value;
	}
}

template <bool RL, int blockSize>
__global__ void refineSubPixel(float *d_DMI, int *d_Cost, unsigned int *d_Confidence, unsigned int *d_DSI, 
							   int disp_start, int disp_stop, int disp_step, int maxCosts,
							   int iWidth, int iHeight) {
	const int disparities = (disp_stop - disp_start + 1) / disp_step;
	int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	int minDisp, minCost, curCost;
	int predecessor, successor;
	int d;
	int offset1, offset2;
	float subPixelDisp;
	int disp_cut;

	if (x < iWidth && y < iHeight) {
		minDisp = 0;
		minCost = maxCosts;

		offset1 = IMUL(iWidth, iHeight);
		offset2 = IMUL(y, iWidth);

		if (x == 400 && y == 49)
			x+=0;

		if (RL) {
			for (d = 0, disp_cut = disp_start+disp_step*d; d < disparities && x < iWidth-disp_cut; d++, disp_cut+=disp_step) {
			//for (d = 0; d < disparities; d++) {
				//disp_cut = disp_start+disp_step*d;

				//if (x < iWidth-disp_cut) {
					// load w/o texture, because loading coalesces
					curCost = d_DSI[d*offset1 + offset2 + x];
					//curCost = tex1Dfetch(texDSI1d, d*offset1 + offset2 + x);

					if (curCost <= minCost) {
						minDisp = d;
						minCost = curCost;
					}
				//}
			}
		}
		else {
			//for (d = 0, disp_cut = disp_start+disp_step*d; d < disparities && x >= disp_cut; d++, disp_cut+=disp_step) {
			for (d = 0, disp_cut = disp_start+disp_step*d; d < disparities && x >= disp_cut; d++, disp_cut+=disp_step) {
				//disp_cut = disp_start+disp_step*d;

				//if (x >= disp_cut) {
					//curCost = d_DSI[d*offset1 + offset2 + x-disp_cut];
					curCost = tex1Dfetch(texDSI1d, d*offset1 + offset2 + x-disp_cut);

					if (curCost <= minCost) {
						minDisp = d;
						minCost = curCost;
					}
				//}
			}
		}



		if (minDisp > 0 && minDisp < disp_stop && (x != disp_cut-1)) {
			if (RL) {
				//predecessor = getArrayValue(d_DSI, x, y, minDisp-1, iWidth, iHeight);
				//successor = getArrayValue(d_DSI, x, y, minDisp+1, iWidth, iHeight);
				predecessor = tex1Dfetch(texDSI1d, (minDisp-1)*offset1 + offset2 + x);
				successor = tex1Dfetch(texDSI1d, (minDisp+1)*offset1 + offset2 + x);
			}
			else {
				//predecessor = getArrayValue(d_DSI, x-disp_start-(minDisp-1)*disp_step, y, minDisp-1, iWidth, iHeight);
				//successor = getArrayValue(d_DSI, x-disp_start-(minDisp+1)*disp_step, y, minDisp+1, iWidth, iHeight);
				predecessor = tex1Dfetch(texDSI1d, (minDisp-1)*offset1 + offset2 + x-disp_start-(minDisp-1)*disp_step);
				successor = tex1Dfetch(texDSI1d, (minDisp+1)*offset1 + offset2 + x-disp_start-(minDisp+1)*disp_step);
			}

			if (2*minCost-predecessor-successor == 0)
				subPixelDisp = (minDisp * disp_step + disp_start);
			else {
				subPixelDisp = (minDisp * disp_step + disp_start) + ((successor-predecessor) / (float)(2*(2*minCost-predecessor-successor)));

				if (subPixelDisp > disp_stop)
					subPixelDisp = disp_stop;
				else if (subPixelDisp < disp_start)
					subPixelDisp = disp_start;
			}
		}
		else {
			subPixelDisp = (minDisp * disp_step + disp_start);
		}

/*
		// Confidence
		int minDispConf = 0;
		int minCostConf = maxCosts;
		if (RL) {
			for (d = 0, disp_cut = disp_start+disp_step*d; d < disparities && x < iWidth-disp_cut; d++, disp_cut+=disp_step) {
				curCost = d_DSI[d*offset1 + offset2 + x];

				if (curCost <= minCostConf && abs(minDisp - d) > 2) {
					minCostConf = curCost;
				}
			}
		}
		else {
			for (d = 0, disp_cut = disp_start+disp_step*d; d < disparities && x >= disp_cut; d++, disp_cut+=disp_step) {
				curCost = tex1Dfetch(texDSI1d, d*offset1 + offset2 + x-disp_cut);

				if (curCost <= minCostConf && abs(minDisp - d) > 2) {
					minCostConf = curCost;
				}
			}
		}
		int conf = 1024*(abs(minCostConf-minCost)/(float)maxCosts);
		if (conf > 255)
			conf = 255;*/


		if (!RL) {
			int minDispConf = 0;
			int minCostConf = maxCosts;
				for (d = 0, disp_cut = disp_start+disp_step*d; d < disparities && x >= disp_cut; d++, disp_cut+=disp_step) {
					curCost = tex1Dfetch(texDSI1d, d*offset1 + offset2 + x-disp_cut);

					if (curCost <= minCostConf && abs(minDisp - d) > 2) {
						minCostConf = curCost;
					}
				}
			unsigned int conf = 1024*(abs(minCostConf-minCost)/(float)maxCosts);
			if (conf > 255)
				conf = 255;

			setArrayValue(d_Confidence, conf, x, y, iWidth);

		}

		setArrayValue(d_DMI, subPixelDisp, x, y, iWidth);
		setArrayValue(d_Cost, minCost, x, y, iWidth);
		//setArrayValue(d_Confidence, conf, x, y, iWidth);
	}
}

__global__ void compareDispsLeft(float *d_DMI, int *d_Costs, unsigned int *d_Confidence, 
								 float *d_DMI_LR, float *d_DMI_RL, 
								 int *d_CostsLR, int *d_CostsRL,
								 unsigned int *d_ConfidenceLR, unsigned int *d_ConfidenceRL,
								 float max_disp_diff, int maxCosts, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;

	float a, b, diff;

	if (x == 400 && y == 49)
		a = 5;
	if (x < iWidth && y < iHeight) {
		a = getArrayValue(d_DMI_LR, x, y, iWidth);
		if (a > x || x-a >= iWidth) b = a; // for debuging only
		else b = getArrayValue(d_DMI_RL, f2i(x-a), y, iWidth);
		diff = abs(a - b);

		/*
		if (diff == a) {
			setArrayValue(d_DMI, a, x, y, iWidth);
			setArrayValue(d_Costs, getArrayValue(d_CostsLR, x, y, iWidth), x, y, iWidth);
		}
		else if (diff == b) {
			if (x+b < iWidth) {
				setArrayValue(d_DMI, b, f2i(x+b), y, iWidth);
				setArrayValue(d_Costs, getArrayValue(d_CostsRL, f2i(x+b), y, iWidth), f2i(x+b), y, iWidth);
			}
		}
		else */if (diff <= max_disp_diff) {
			setArrayValue(d_DMI, (a+b)/2, x, y, iWidth);
			setArrayValue(d_Costs, getArrayValue(d_CostsLR, x, y, iWidth), x, y, iWidth);
			setArrayValue(d_Confidence, getArrayValue(d_ConfidenceLR, x, y, iWidth), x, y, iWidth);
		}
		else {
			setArrayValue(d_DMI, 0.0f, x, y, iWidth);
			setArrayValue(d_Costs, maxCosts, x, y, iWidth);
			setArrayValue(d_Confidence, 0u, x, y, iWidth);
		}
	}
}

__global__ void thresholdConfidence(float *d_DMI, unsigned int *d_Confidence, int threshold, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	
	unsigned int value;
		
	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_Confidence, x, y, iWidth);

		if (value <= threshold) {
			setArrayValue(d_DMI, 0.f, x, y, iWidth);
		}
	}
}

__global__ void thresholdTexture(float *d_DMI, unsigned int *d_Texture, int threshold, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	
	int value;
		
	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_Texture, x, y, iWidth);

		if (value <= threshold) {
			setArrayValue(d_DMI, 0.f, x, y, iWidth);
		}
	}
}

__global__ void roundDisparities(float *d_DMI, int resolution, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		value *= resolution;
		value += 0.5f;
		value = (float)((int)value) / resolution;

		setArrayValue(d_DMI, value, x, y, iWidth);
	}
}

__global__ void scaleDisparities(float *d_DMI, int scale, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		value *= scale;

		setArrayValue(d_DMI, value, x, y, iWidth);
	}
}

__global__ void roundAndScaleDisparities(float *d_DMI, int resolution, int scale, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		// round
		value *= resolution;
		value += 0.5f;
		value = (float)((int)value) / resolution;

		// scale
		value *= scale;

		setArrayValue(d_DMI, value, x, y, iWidth);
	}
}


__global__ void roundScaleThresholdDisparities(float *d_DMI, unsigned int *d_Confidence, unsigned int *d_Texture, int thresholdConfidence, 
											   int thresholdTexture, int resolution, int scale, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;
	unsigned int texValue, confValue;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		texValue = getArrayValue(d_Texture, x, y, iWidth);
		confValue = getArrayValue(d_Confidence, x, y, iWidth);

		if (texValue <= thresholdTexture || confValue <= thresholdConfidence)
			value = 0.0f;

		// round
		value *= resolution;
		value += 0.5f;
		value = (float)((int)value) / resolution;

		// scale
		value *= scale;

		setArrayValue(d_DMI, value, x, y, iWidth);
	}
}



__global__ void calcDepthMap(unsigned int *d_DepthMap, float *d_DMI, float f, float b, int iWidth, int iHeight) {
	const int x = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int y = IMUL(blockDim.y, blockIdx.y) + threadIdx.y;
	float value;

	if (x < iWidth && y < iHeight) {
		value = getArrayValue(d_DMI, x, y, iWidth);

		value = (256*f*b)/value;

		setArrayValue(d_DepthMap, (unsigned int)value, x, y, iWidth);
	}
}


// takes about 1.84 ms for a 16x16 census
extern "C" void gpuCensusTransform() {

	hipStreamSynchronize( streamImageLeft[g_streamNr] ) ;
	hipStreamSynchronize( streamImageRight[g_streamNr] ) ;

	START_TIMER;

	if (g_sparse) {
#define CENSUS_SM
#ifdef CENSUS_SM
		dim3 grid(iDivUp(g_Width, CENSUS_SPARSE_TILE_W), iDivUp(g_Height, CENSUS_SPARSE_TILE_H));
		dim3 block(16, 16);

		// left
		censusTransformSparse<true> <<<grid, block>>>(d_censusLeft, g_Width, g_Height);

		// right
		censusTransformSparse<false> <<<grid, block>>>(d_censusRight, g_Width, g_Height);
#else
		dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
		dim3 block(16, 16);

		// left
		censusTransformSparseTex<true> <<<grid, block>>>(d_censusLeft, g_Width, g_Height);

		// right
		censusTransformSparseTex<false> <<<grid, block>>>(d_censusRight, g_Width, g_Height);
#endif

//		printf("  Bandwidth: %.1f GB/s\n", 2*(iDivUp(g_Width, CENSUS_SPARSE_TILE_W)*iDivUp(g_Height, CENSUS_SPARSE_TILE_H)*CENSUS_SPARSE_SMEM_W*CENSUS_SMEM_H*sizeof(unsigned int)+g_Width*g_Height*8)/(gpuTime*1000000));
//		printf("  GFLOPS:  : %.1f\n", 2*3000*16*16*iDivUp(g_Width, 64)*iDivUp(g_Height, 64)/(gpuTime*1000000));
	}
	else {
		dim3 grid(iDivUp(g_Width, CENSUS_TILE_W), iDivUp(g_Height, CENSUS_TILE_H));
		dim3 block(8, 24);

		// left
		censusTransform<true> <<<grid, block>>>(d_censusLeft, g_Width, g_Height);

		// right
		censusTransform<false> <<<grid, block>>>(d_censusRight, g_Width, g_Height);
	}

	STOP_TIMER(g_CensusTiming[eCensusTransform]);
}

extern "C" void gpuCalcDSI() {
	unsigned int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;
	int sharedMem;

	START_TIMER;

	// ((2 * (#threads + maxdisparity)) * nrOfInt) int values must be allocated
	if (g_sparse) {
		dim3 grid(iDivUp(g_Width, 128), g_Height);
		dim3 block(128, 1);

		//sharedMem = 2 * (128 + disparities) * 2 * sizeof(unsigned int);
		sharedMem = (2 * 128 + disparities) * 2 * sizeof(unsigned int);

#if USE_FAST_AGGREGATION
		calcDSI<5, 2> <<<grid, block, sharedMem>>>(d_DSI_Temp, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
#else
		calcDSI<2> <<<grid, block, sharedMem>>>(d_DSI, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
#endif
	}
	else {
//		dim3 grid(iDivUp(g_Width, 128), g_Height);
//		dim3 block(128, 1);
//
//		sharedMem = 2 * (128 + disparities) * 8 * sizeof(unsigned int);
//
//#if USE_FAST_AGGREGATION
//		calcDSI<8> <<<grid, block, sharedMem>>>(d_DSI_Temp, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
//#else
//		calcDSI<8> <<<grid, block, sharedMem>>>(d_DSI, d_censusLeft, d_censusRight, g_disp_min, g_disp_max, g_disp_step, g_Width, g_Height);
//#endif	
	}

	STOP_TIMER(g_CensusTiming[eCalcDSI]);

//	printf("  Bandwidth: %.1f GB/s\n", (iDivUp(g_Width, 128)*g_Height*2*(128 + disparities)*2*sizeof(unsigned int)+2*(g_Width-g_disp_max)*g_Height*sizeof(unsigned int))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", (5*(128+disparities)*2*g_Height*iDivUp(g_Width, 128)+(17+43*disparities)*g_Width*g_Height)/(gpuTime*1000000));
}

extern "C" void gpuAggregateCosts() {
	int disparities = (g_disp_max + 1 - g_disp_min) / g_disp_step;
	int d;

	//dim3 grid(iDivUp(g_Width, 16), iDivUp((g_Height+g_blockSize)*disparities, 16));
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

    dim3 blockGridRows(iDivUp(g_Width, ROW_TILE_W), g_Height);
    dim3 blockGridColumns(iDivUp(g_Width, COLUMN_TILE_W), iDivUp(g_Height, COLUMN_TILE_H));
    dim3 threadBlockRows(KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS);
    dim3 threadBlockColumns(COLUMN_TILE_W, 8);

	START_TIMER;

#if USE_FAST_AGGREGATION



	for (d = 0; d < disparities; d++) {
        //convolutionRowGPU<<<blockGridRows, threadBlockRows>>>(
        //    &d_DSI_Temp2[g_Width*g_Height*d],
        //    &d_DSI_Temp[g_Width*g_Height*d],
        //    g_Width,
        //    g_Height
        //);

        //convolutionColumnGPU<<<blockGridColumns, threadBlockColumns>>>(
        //    &d_DSI[g_Width*g_Height*d],
        //    &d_DSI_Temp2[g_Width*g_Height*d],
        //    g_Width,
        //    g_Height,
        //    COLUMN_TILE_W * threadBlockColumns.y,
        //    g_Height * threadBlockColumns.y
        //);

		aggregateFilter<false, AGGFILTERWIDTH> <<<grid, block>>> (&d_DSI[g_Width*g_Height*d], &d_DSI_Temp[g_Width*g_Height*d], g_Width, g_Height);
		//aggregateFilter<false, 5> <<<grid, block>>> (&d_DSI[g_Width*g_Height*d], &d_DSI_Temp[(g_Width+g_blockSize)*(g_Height+g_blockSize)*d], g_Width, g_Height);
		//aggregateFilter<true, 5> <<<grid, block>>> (d_DSI, d_DSI_Temp, g_Width, (g_Height+g_blockSize)*disparities-g_blockSize);
	}
#else
	unsigned int *d_Temp;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Temp, g_Width * g_Height * sizeof(unsigned int)));

	for (d = 0; d < disparities; d++) {
		CUDA_SAFE_CALL( hipMemcpy(d_Temp, &d_DSI[g_Width*g_Height*d], g_Width*g_Height*sizeof(unsigned int), hipMemcpyDeviceToDevice) );
		aggregateFilter<false, filterWidth> <<<grid, block>>> (&d_DSI[g_Width*g_Height*d], d_Temp, g_Width, g_Height);
	}

	CUDA_SAFE_CALL( hipFree(d_Temp) );
#endif

	STOP_TIMER(g_CensusTiming[eAggregateCosts]);

//	printf("  Bandwidth: %.1f GB/s\n", disparities*(g_Width*g_Height*sizeof(int) + iDivUp(g_Width, 16)*iDivUp(g_Height, 16)*21*21*sizeof(int))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", disparities*60*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuRefineSubPixel() {
	/*
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 8));
	dim3 block(16, 8);
	*/
	dim3 grid(iDivUp(g_Width, 128), iDivUp(g_Height, 1));
	dim3 block(128, 1);

	START_TIMER;

	refineSubPixel<true, 5> <<<grid, block>>>(d_DMI_RL, d_Costs_RL, d_Confidence_RL, d_DSI, g_disp_min, g_disp_max, g_disp_step, 1600, g_Width, g_Height);

	//printf("  Bandwidth: %.1f GB/s\n", ((disparities+2)*g_Width*g_Height*sizeof(int))/(gpuTime*1000000));
	//printf("  GFLOPS:  : %.1f\n", ((disparities*7+35)*g_Width*g_Height) /(gpuTime*1000000) );

	refineSubPixel<false, 5> <<<grid, block>>>(d_DMI_LR, d_Costs_LR, d_Confidence_LR, d_DSI, g_disp_min, g_disp_max, g_disp_step, 1600, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eRefineSubPixel]);

	//printf("  Bandwidth: %.1f GB/s\n", ((disparities+2)*g_Width*g_Height*sizeof(int))/(gpuTime*1000000));
	//printf("  GFLOPS:  : %.1f\n", ((disparities*7+43)*g_Width*g_Height) /(gpuTime*1000000) );
}

extern "C" void gpuCalcTextureMap() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	calcTextureMap<<<grid, block>>>(d_Texture, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eCalcTextureMap]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuCompareDisps() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	compareDispsLeft<<<grid, block>>>(d_DMI, d_Costs, d_Confidence, d_DMI_LR, d_DMI_RL, d_Costs_LR, d_Costs_RL, d_Confidence_LR, d_Confidence_RL, 1.0, 1600, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eCompareDisps]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuRoundAndScaleDisparities() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	roundAndScaleDisparities<<<grid, block>>>(d_DMI, 10, g_dmScale, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eRoundAndScaleDisparities]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 12*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuRoundScaleThresholdDisparities() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	roundScaleThresholdDisparities<<<grid, block>>>(d_DMI, d_Confidence, d_Texture, 10, 15, 10, g_dmScale, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eRoundAndScaleDisparities]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 12*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuThresholdConfidence() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	thresholdConfidence<<<grid, block>>>(d_DMI, d_Confidence, 20, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eThresholdConfidence]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuCalcDepthMap(float f, float b) {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	calcDepthMap<<<grid, block>>>(d_DepthMap, d_DMI, f, b, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eCalcDepthMap]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}

extern "C" void gpuThresholdTexture() {
	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	START_TIMER;

	thresholdTexture<<<grid, block>>>(d_DMI, d_Texture, 15, g_Width, g_Height);

	STOP_TIMER(g_CensusTiming[eThresholdTexture]);

//	printf("  Bandwidth: %.1f GB/s\n", (2*g_Width*g_Height*sizeof(float))/(gpuTime*1000000));
//	printf("  GFLOPS:  : %.1f\n", 16*16*16*iDivUp(g_Width, 16)*iDivUp(g_Height, 16) /(gpuTime*1000000) );
}


extern "C" void gpuCensusImageSetup(unsigned int w, unsigned int h, unsigned int disp_min, unsigned int disp_max, unsigned int disp_step, bool sparse, unsigned int blockSize, int dmScale) {
	unsigned int disparities = (disp_max + 1 - disp_min) / disp_step;

	g_Width = w;
	g_Height = h;

	g_disp_min = disp_min;
	g_disp_max = disp_max;
	g_disp_step = disp_step;
	g_sparse = sparse;
	g_blockSize = blockSize;
	g_dmScale = dmScale;

	for (int i=0; i<2; i++) {
		CUDA_SAFE_CALL( hipStreamCreate(&streamImageLeft[i]) ); 
		CUDA_SAFE_CALL( hipStreamCreate(&streamImageRight[i]) ); 
		CUDA_SAFE_CALL( hipStreamCreate(&streamDM[i]) ); 

		CUDA_SAFE_CALL( hipHostMalloc((void**)&h_left[i],  g_Width*g_Height*sizeof(unsigned char)) );
		CUDA_SAFE_CALL( hipHostMalloc((void**)&h_right[i], g_Width*g_Height*sizeof(unsigned char)) );
		CUDA_SAFE_CALL( hipHostMalloc((void**)&h_dm[i],    g_Width*g_Height*sizeof(unsigned char)) );
	}

	hipChannelFormatDesc channelDescChar = hipCreateChannelDesc<unsigned char>();
	hipChannelFormatDesc channelDescInt = hipCreateChannelDesc<unsigned int>();

	CUDA_SAFE_CALL(hipMallocArray(&cuImageLeft, &channelDescChar, g_Width, g_Height));
	CUDA_SAFE_CALL(hipMallocArray(&cuImageRight, &channelDescChar, g_Width, g_Height));
	CUDA_SAFE_CALL(hipBindTextureToArray(texImageLeft,  cuImageLeft, channelDescChar));
	CUDA_SAFE_CALL(hipBindTextureToArray(texImageRight, cuImageRight, channelDescChar));

	if (sparse) {
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusLeft, g_Width*g_Height*8));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusRight, g_Width*g_Height*8));
	}
	else {
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusLeft, g_Width*g_Height*256/8));
		CUDA_SAFE_CALL(hipMalloc((void**)&d_censusRight, g_Width*g_Height*256/8));
	}

	CUDA_SAFE_CALL(hipMalloc((void**)&d_DSI, disparities * (g_Width+blockSize) * (g_Height+blockSize) * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipBindTexture(0, texDSI1d, d_DSI, disparities * (g_Width+blockSize) * (g_Height+blockSize) * sizeof(unsigned int)));

#if USE_FAST_AGGREGATION
	CUDA_SAFE_CALL(hipMalloc((void**)&d_DSI_Temp, disparities * (g_Width+blockSize) * (g_Height+blockSize) * sizeof(unsigned int)));
	//CUDA_SAFE_CALL(hipMalloc((void**)&d_DSI_Temp2, disparities * (g_Width+blockSize) * (g_Height+blockSize) * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipBindTexture(0, texDSITemp1d, d_DSI_Temp, disparities * (g_Width+blockSize) * (g_Height+blockSize) * sizeof(unsigned int)));
#endif

	//CUDA_SAFE_CALL(hipMalloc((void**)&d_integralImage, ((g_Width-disp_max) + 2 * g_blockSize) * (g_Height + 2 * g_blockSize) * sizeof(unsigned int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI, g_Width * g_Height * sizeof(float)));
//	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI2, g_Width * g_Height * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI_RL, g_Width * g_Height * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_DMI_LR, g_Width * g_Height * sizeof(float)));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_Texture, g_Width * g_Height * sizeof(unsigned int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_DepthMap, g_Width * g_Height * sizeof(unsigned int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_Confidence, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Confidence_LR, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Confidence_RL, g_Width * g_Height * sizeof(int)));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_Costs, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Costs_RL, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_Costs_LR, g_Width * g_Height * sizeof(int)));

	//CUDA_SAFE_CALL(hipMemset(d_integralImage, 0, ((g_Width-disp_max) + 2 * g_blockSize) * (g_Height + 2 * g_blockSize) * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemset(d_DSI, 0, disparities * g_Width * g_Height * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemset(d_DSI_Temp, 0, disparities * (g_Width+blockSize) * (g_Height+blockSize) * sizeof(unsigned int)));
	//CUDA_SAFE_CALL(hipMemset(d_DSI_Temp2, 0, disparities * (g_Width+blockSize) * (g_Height+blockSize) * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemset(d_Costs, 0, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_Costs_RL, 0, g_Width * g_Height * sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(d_Costs_LR, 0, g_Width * g_Height * sizeof(int)));

	dim3 grid(iDivUp(g_Width, 16), iDivUp(g_Height, 16));
	dim3 block(16, 16);

	setArray<<<grid, block>>>(d_DMI, 0.f, g_Width, g_Height);
	setArray<<<grid, block>>>(d_DMI_RL, -1.f, g_Width, g_Height);
	setArray<<<grid, block>>>(d_DMI_LR, -1.f, g_Width, g_Height);
}

extern "C" void gpuCensusImageCleanup() {
	CUDA_SAFE_CALL( hipUnbindTexture(texImageLeft) );
	CUDA_SAFE_CALL( hipUnbindTexture(texImageRight) );

	for (int i=0; i<2; i++) {
		CUDA_SAFE_CALL( hipStreamDestroy(streamImageLeft[i]) ); 
		CUDA_SAFE_CALL( hipStreamDestroy(streamImageRight[i]) ); 
		CUDA_SAFE_CALL( hipStreamDestroy(streamDM[i]) ); 

		CUDA_SAFE_CALL( hipHostFree(h_left[i]) );
		CUDA_SAFE_CALL( hipHostFree(h_right[i]) );
		CUDA_SAFE_CALL( hipHostFree(h_dm[i]) );
	}

	CUDA_SAFE_CALL( hipFreeArray(cuImageLeft) );
	CUDA_SAFE_CALL( hipFreeArray(cuImageRight) );

    CUDA_SAFE_CALL( hipFree(d_censusLeft) );
    CUDA_SAFE_CALL( hipFree(d_censusRight) );
    CUDA_SAFE_CALL( hipFree(d_DSI) );
#if USE_FAST_AGGREGATION
    CUDA_SAFE_CALL( hipFree(d_DSI_Temp) );
//    CUDA_SAFE_CALL( hipFree(d_DSI_Temp2) );
#endif
    CUDA_SAFE_CALL( hipFree(d_DMI) );
//    CUDA_SAFE_CALL( hipFree(d_DMI2) );
    CUDA_SAFE_CALL( hipFree(d_DMI_LR) );
    CUDA_SAFE_CALL( hipFree(d_DMI_RL) );
    CUDA_SAFE_CALL( hipFree(d_Texture) );
    CUDA_SAFE_CALL( hipFree(d_DepthMap) );
    CUDA_SAFE_CALL( hipFree(d_Confidence) );
    CUDA_SAFE_CALL( hipFree(d_Confidence_LR) );
    CUDA_SAFE_CALL( hipFree(d_Confidence_RL) );
    CUDA_SAFE_CALL( hipFree(d_Costs) );
    CUDA_SAFE_CALL( hipFree(d_Costs_LR) );
    CUDA_SAFE_CALL( hipFree(d_Costs_RL) );
}

extern "C" unsigned char* gpuGetLeftImageBuffer(int nr) {
	if (nr == 0 || nr == 1)
		return h_left[nr];
	else
		return NULL;
}

extern "C" unsigned char* gpuGetRightImageBuffer(int nr) {
	if (nr == 0 || nr == 1)
		return h_right[nr];
	else
		return NULL;
}

extern "C" void gpuCensusSetAsyncImageNr(int nr) {
	g_streamNr = nr;
}

extern "C" void gpuCensusLoadImages(int nr) {
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageLeft, 0, 0, h_left[nr], DATA_SIZE, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageRight, 0, 0, h_right[nr], DATA_SIZE, hipMemcpyHostToDevice) );
}

extern "C" void gpuCensusSetImages(unsigned char *left, unsigned char *right) {
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageLeft, 0, 0, left, DATA_SIZE, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpyToArray(cuImageRight, 0, 0, right, DATA_SIZE, hipMemcpyHostToDevice) );
}

extern "C" void gpuGetDisparityMap(float *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_DMI, g_Width*g_Height*sizeof(float), hipMemcpyDeviceToHost) );
}

extern "C" void gpuGetConfidenceMap(int *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_Confidence, g_Width*g_Height*sizeof(int), hipMemcpyDeviceToHost) );
}

extern "C" void gpuGetTexture(int *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_Texture, g_Width*g_Height*sizeof(int), hipMemcpyDeviceToHost) );
}

extern "C" void gpuGetDepthMap(int *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_DepthMap, g_Width*g_Height*sizeof(int), hipMemcpyDeviceToHost) );
}

extern "C" void debugGetCensusLeft(int *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_censusLeft, g_Width*g_Height*sizeof(int)*2, hipMemcpyDeviceToHost) );
}
extern "C" void debugGetCensusRight(int *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_censusRight, g_Width*g_Height*sizeof(int)*8, hipMemcpyDeviceToHost) );
}
extern "C" void debugGetDSI(int *h_data, int d) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, &d_DSI[g_Width*g_Height*d], g_Width*g_Height*sizeof(int), hipMemcpyDeviceToHost) );
}
extern "C" void debugGetDMI_LR(float *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_DMI_LR, g_Width*g_Height*sizeof(float), hipMemcpyDeviceToHost) );
}
extern "C" void debugGetDMI_RL(float *h_data) {
	CUDA_SAFE_CALL( hipMemcpy(h_data, d_DMI_RL, g_Width*g_Height*sizeof(float), hipMemcpyDeviceToHost) );
}
