// vim:set fileencoding=utf-8 sw=3 ts=3 et:vim
/*
 * @author:  Marko Mahnič
 * @created: jul 2009 
 */
   

#include <stdio.h>
#include <string.h>
#include <cutil.h>

#undef VERBOSE

#include "cudaImage.h"
#include "cudaSift.h"

void UploadSiftData(SiftData *data)
{
   if (data->h_data == NULL) return; // No data to upload to device
   if (data->maxPts < 1 || data->numPts < 1) return;
   if (data->d_data == NULL) {
      int sz = sizeof(SiftPoint)*data->maxPts;
      printf("allocating %d, %d, ", data->maxPts, sz);
      CUDA_SAFE_CALL(hipMalloc((void **)&data->d_data, sz));
   }
   CUDA_SAFE_CALL(hipMemcpy(data->d_data, data->h_data, 
      sizeof(SiftPoint)*data->numPts, hipMemcpyHostToDevice));
}
